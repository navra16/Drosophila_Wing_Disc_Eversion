#include "hip/hip_runtime.h"
#include <stdio.h>
#include "System.h"
#include "SystemStructures.h" 
#include "AreaTriangles.h"
//#include "AreaTrianglesEnergy.h"
#include "BendingTriangles.h"
// #include "BendingTrianglesEnergy.h"
#include "MemRepulsionSprings_universal.h"
#include "MemRepulsionSprings_local.h"
#include "MemRepulsionEnergy.h"
#include "LinearSprings.h"
//#include "LinearSpringsEnergy.h"
//#include "LJSprings.h"
//#include "LJSprings_LJ.h"
#include "NodeAdvance.h"
//#include "BucketScheme.h"
#include "Storage.h" 
#include "Utilities.h"
#include "SystemBuilder.h"
#include <vector>
#include "VolumeComp.h"
#include "VolumeSprings.h"
#include <bits/stdc++.h>
#include "LineTensionSprings.h"
//#include "Growth.h"
#include <math.h>
#include "vmalloc.h"
#include <list>
#include "TurgorForce.h"

//#include "SurfaceNormal.h"
//#include "Nodes2Triangles.h"



 //somehow the gradient is not being set in my version

//bool IsPos (int i){return (i>=0);}
int count_bigger(const std::vector<int>& elems) {
    return std::count_if(elems.begin(), elems.end(), [](int c){return c >= 0;});
}

System::System() {};
void System::Solve_Forces(){

	thrust::fill(coordInfoVecs.nodeForceX.begin(), coordInfoVecs.nodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceY.begin(), coordInfoVecs.nodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceZ.begin(), coordInfoVecs.nodeForceZ.end(), 0.0);
	
	//setBucketScheme();
	ComputeLinearSprings(
		generalParams, 
		coordInfoVecs,
		linearSpringInfoVecs, 
		ljInfoVecs);
	// std::cout<<"ERROR 1"<<std::endl;
	ComputeAreaTriangleSprings(
		
		generalParams,
		coordInfoVecs,
		areaTriangleInfoVecs);
		// std::cout<<"ERROR 2"<<std::endl;
	ComputeTurgorSprings(
		generalParams,
		coordInfoVecs,
		areaTriangleInfoVecs
	);
	// std::cout<<"ERROR 3"<<std::endl;
	ComputeCosTriangleSprings(
		
		generalParams,
		coordInfoVecs,  
		bendingTriangleInfoVecs); 
		// std::cout<<"ERROR 4"<<std::endl;
	// ComputeMemRepulsionSprings_universal(
	// 	coordInfoVecs,
	// 	linearSpringInfoVecs, 
	// 	capsidInfoVecs,
	// 	generalParams,
	// 	auxVecs);

	ComputeMemRepulsionSprings_local(
		coordInfoVecs,
		linearSpringInfoVecs, 
		capsidInfoVecs,
		generalParams,
		auxVecs);
		// std::cout<<"ERROR 5"<<std::endl;

	ComputeVolume(
		generalParams,
		coordInfoVecs,
		linearSpringInfoVecs,
		ljInfoVecs);

		// std::cout<<"ERROR 6"<<std::endl;
	/*ComputeVolumeSprings(
		coordInfoVecs,
		linearSpringInfoVecs, 
		capsidInfoVecs,
		generalParams,
		auxVecs);*/

	/* if (generalParams.true_current_total_volume/initial_volume >= 1.25){
	ComputeLineTensionSprings(
		generalParams,
		coordInfoVecs,
		linearSpringInfoVecs);
	} */
		
};

void System::solveSystem(){

	coordInfoVecs.k_0 = 20.0;
	coordInfoVecs.k_1 = 25.0;
	coordInfoVecs.k_2 = 5.0;
	coordInfoVecs.k_3 = 5.0;
	coordInfoVecs.k_4 = 1.0;
	coordInfoVecs.k_ss = 12;//10.75;
	coordInfoVecs.beta = 1.0/1.0;///1.45;
	coordInfoVecs.gamma = 1.0;
	coordInfoVecs.q1 = 10.0;
	coordInfoVecs.h = 10.0;

	uint mem_prealloc = 4; //Make sure that this number is the same as set in System::initializeSystem found near the bottom of this script.

	double max_conc_scaler_for_material_insert = 0.8;
	std::cout<<"multiplier applied to max conc to determine location of material insertion = "<<max_conc_scaler_for_material_insert<<std::endl;
	double current_edge_to_tip_height_scale = 2.0;
	std::cout<<"current_edge_to_tip_height_scale = "<<current_edge_to_tip_height_scale<<std::endl;
	//Determines how far away from the tip can new material be inserted.
	// double current_edge_to_tip_dist_scale = 4.0;
	// std::cout<<"current_edge_to_tip_dist_scale = "<<current_edge_to_tip_dist_scale<<std::endl;
	double bdry_to_tip_height_scale = 0.0;
	std::cout<<"bdry_to_tip_height_scale = "<<bdry_to_tip_height_scale<<std::endl;


	// double u_scalingPower = 8;//6;//8;//4;//6.0;
	double u_scalingPower = 7; // For test-compilation purpose, please delete this line and reactive the line above.
	std::cout<<"u_scalingPower (power of 'u' in the chem diff) = "<<u_scalingPower<<std::endl;
	double max_u_scalingPower = 8;
	std::cout<<"max u_scalingPower = "<<max_u_scalingPower<<std::endl;
	double max_u_scalingPower_postPeak = 8;
	std::cout<<"max_u_scalingPower_postPeak = "<<max_u_scalingPower_postPeak<<std::endl;
	double timeFrame = 5e3;
	std::cout<<"timeFrame needed to reach max u_scalingPower = "<<timeFrame<<std::endl;
	double timeFrame_postPeak = 5e3;
	std::cout<<"timeFrame needed to reach max u_scalingPower_postPeak = "<<timeFrame_postPeak<<std::endl;
	double u_scalingPower_Progress = sqrt((max_u_scalingPower - u_scalingPower)*(max_u_scalingPower - u_scalingPower))/timeFrame;
	double u_scalingPower_postPeak_Progress = sqrt((max_u_scalingPower - max_u_scalingPower_postPeak)*(max_u_scalingPower - max_u_scalingPower_postPeak))/timeFrame_postPeak;
	bool polarization_postPeak = false;
	bool triggered = false;
	generalParams.current_total_sim_step = 0;
	int relax_max_steps_before_growth_and_edgeswap = 3e3;
	Front *front = new Front();
	RBC *rbc = new RBC();
	RBC *n_rbc = new RBC();
	auto utilities_ptr = std::make_shared<Utilities>(coordInfoVecs, generalParams);
    auto build_ptr = weak_bld_ptr.lock();
	std::cout<<"Declaration of rbc and n_rbc complete."<<std::endl;
	std::cout<<"Utilities_ptr declaration complete."<<std::endl;
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////Build the "u" vector representing the external or internal influencer for polarization /////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	double max_cell_triangle_height, min_cell_triangle_height, v1,v2,v3,v4, cell_height;
	max_cell_triangle_height = -10000.0;
	min_cell_triangle_height = 10000.0;

	for (int i = 0; i < coordInfoVecs.num_triangles; i++){
		if (coordInfoVecs.triangles2Nodes_1[i] >= (INT_MAX-100) || coordInfoVecs.triangles2Nodes_2[i] >= (INT_MAX-100) || coordInfoVecs.triangles2Nodes_3[i] >= (INT_MAX-100)){
			continue;
		}
		if (coordInfoVecs.triangles2Nodes_1[i] <= (-INT_MAX+100) || coordInfoVecs.triangles2Nodes_2[i] <= (-INT_MAX+100) || coordInfoVecs.triangles2Nodes_3[i] <= (-INT_MAX+100)){
			continue;
		}
		v1 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[i]];
		v2 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[i]];
		v3 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[i]];
		v4 = (v1+v2+v3)/3.0;
		if (v4 >= max_cell_triangle_height){
			max_cell_triangle_height = v4;
		}
		if (v4 <= min_cell_triangle_height){
			min_cell_triangle_height = v4;
		}
	}
	cell_height = (max_cell_triangle_height - min_cell_triangle_height);
	std::cout<<"Determination of max triangle and min cell height complete."<<std::endl;
	double max_u = 1.5;//1.1;
	double min_u = 0.5;//0.9;
	std::cout<<"max_u = "<<max_u<<std::endl;
	std::cout<<"min_u = "<<min_u<<std::endl;
	// vector<double> u;
	// coordInfoVecs.u.resize(3*coordInfoVecs.num_triangles);
	for (int i = 0; i < coordInfoVecs.num_triangles; i++){
		if (coordInfoVecs.triangles2Nodes_1[i] >= (INT_MAX-100) || coordInfoVecs.triangles2Nodes_2[i] >= (INT_MAX-100) || coordInfoVecs.triangles2Nodes_3[i] >= (INT_MAX-100)){
			continue;
		}
		if (coordInfoVecs.triangles2Nodes_1[i] <= (-INT_MAX+100) || coordInfoVecs.triangles2Nodes_2[i] <= (-INT_MAX+100) || coordInfoVecs.triangles2Nodes_3[i] <= (-INT_MAX+100)){
			continue;
		}
		
		v1 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[i]];
		v2 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[i]];
		v3 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[i]];
		v4 = (v1+v2+v3)/3.0;
		coordInfoVecs.u[i] = min_u + (max_u - min_u)*pow(((v4-min_cell_triangle_height)/cell_height),u_scalingPower); //Try exponential power? C_0*exp(-x/lambda), varying lambda
		// coordInfoVecs.u[i] = max_u - (max_u - min_u)*exp((-(v4-min_cell_triangle_height)/cell_height)/u_scalingPower);
	}
	std::cout<<"Construction of the 'u' vector complete."<<std::endl;
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	generalParams.nodeMass = 1.0;
	int GROWTH_COUNTER = 0;
	int min_num_edge_loop = 1;
	std::cout<<"min_num_edge_loop for edgeswap = "<<min_num_edge_loop<<std::endl;

	// std::random_device rand_dev;
	// // std::mt19937 generator2(rand_dev());
	// std::mt19937 generator_edgeswap(rand_dev());

	double MAX_VOLUME_RATIO = 3.0;
	double MAX_BUD_AREA_RATIO = 100.0;
	int MAX_GROWTH_NUMBER = 1;
	std::cout<<"MAX_GROWTH_NUMBER (# of edge to expand) = "<<MAX_GROWTH_NUMBER<<std::endl;
	int GROWTH_FREQUENCY = 25;//95;//70;//25*3;
	std::cout<<"GROWTH_FREQ (how many times Max_Runtime has to be reached to perform growth"<<GROWTH_FREQUENCY<<std::endl;
	double energy_gradient_threshold = 0.02;//0.01;
	std::cout<<"ENERGY_GRADIENT_THRESHOLD = "<<energy_gradient_threshold<<std::endl;

	generalParams.kT_growth = 1.0;
	generalParams.SCALE_TYPE = 3; 
	// 0:= Gaussian-like weakening
	// 1:= a1*(pow(x,b)) + a2*(1-pow(x,b)) type weakening
	// 2:= pure Gaussian weakening
	// 3:= isotropic
	// 4:= hill equation
	//Note that (3) is used in combination with sigma = INT_MAX;
	std::cout<<"SCALE TYPE = "<<generalParams.SCALE_TYPE<<std::endl;
	std::cout<<"0:= sigmoidal Gaussian-like weakening, 1:= a1*(pow(x,b)) + a2*(1-pow(x,b)) type weakening, 2:= pure Gaussian weakening, 3:= isotropic, 4:= hill equation"<<std::endl;
	generalParams.scaling_pow = 2.0;
	std::cout<<"scaling_pow (this is for SCALE_TYPE = 1 case) = "<<generalParams.scaling_pow<<std::endl;
	generalParams.gausssigma = 0.1;
	std::cout<<"gausssigma (this is for the SCALE_TYPE = 0 case) = "<<generalParams.gausssigma<<std::endl;
	//coordInfoVecs.scaling_per_edge.
	//generalParams.hilleqnconst = 0.9;
	//generalParams.hilleqnpow = 40.0;
	std::vector<int> nodes_in_growth;
	std::vector<int> triangles_in_growth;
	std::vector<int> edges_in_growth;
	double dtb; //dtb := distance to boundary
	double dtb_max; //dtb_max := the max distance used to calculate the distance ratio in the Hill equation.
	double sigma = 0.0;//INT_MAX; //if this is set to be INT_MAX then we assume isotropic weakening.
	double sigma_true = sqrt(0.5); //This is the variance used to calculate the scaling of the wall weakening.
	std::cout<<"initial sigma (for gradient distribution variance), based on initial distribution of Cdc42, if using true gaussian weakening = "<<sigma<<std::endl;
	std::cout<<"If sigma = INT_MAX, then we have isotropic weakening scenario"<<std::endl;
	std::cout<<"true sigma (for gaussian-related distribution variance) = "<<sigma_true<<std::endl;

	generalParams.insertion_energy_cost = -log(0.0025);
	std::cout<<"GROWTH: material insertion energy cost (dependent on local chemical concentration) = "<<generalParams.insertion_energy_cost<<std::endl;
	generalParams.strain_threshold = 0.05;//0.01;
	std::cout<<"GROWTH: critical strain threshold used for insertion probability calculation = "<<generalParams.strain_threshold<<std::endl;

	generalParams.growth_energy_scaling = 1.0;//0.01375;
	std::cout<<"GROWTH ENERGY SCALING FOR MATERIAL INSERTION PROBABILITY = "<<generalParams.growth_energy_scaling<<std::endl;
	generalParams.safeguardthreshold = 9;
	std::cout<<"NEIGHBOR SAFE GUARD THRESHOLD = "<<generalParams.safeguardthreshold<<std::endl;
	//safeguardthreshold is the maximum number of neighboring nodes a node can have.

	//////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////
	////////////////////////// PARAMETER SETTINGS ////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////

	int TOTAL_GROWTH_COUNTER = 0;
	int TOTAL_GROWTH_ATTEMPT = 0;
	double max_runTimePerRelaxation = 50;
	std::cout<<"Number of sim steps per relaxation event : "<<max_runTimePerRelaxation<<std::endl;

	double Max_Runtime = generalParams.dt*max_runTimePerRelaxation;//50.0;//50.0;
	double minimal_run_time_ratio = 1.0;
	double Max_RunStep = Max_Runtime/generalParams.dt;
	std::cout<<"Max runtime = "<<Max_Runtime<<std::endl;
	std::cout<<"Max runstep = "<<Max_RunStep<<std::endl;
	bool runSim = true;
	int num_edge_loop;
	double initial_kT;
	initial_kT = generalParams.kT;//This is for the acceptance of change after looping through every edge within proximity.
	double SAMPLE_SIZE = 0.05;//0.025;
	std::cout<<"Sample ratio: "<<SAMPLE_SIZE<<std::endl;
	std::cout<<"If the Sample raio is 0, it means we have chosen a fixed number of attempt throughout the simulation"<<std::endl;
	//This determines the number of edges to test for bondflip remeshing

	// auto utilities_ptr = std::make_shared<Edgeswap>(coordInfoVecs, generalParams);
	int RECORD_TIME = 1;//round(Max_RunStep/2);
	std::cout<<"Record frequency = "<<RECORD_TIME<<std::endl;
	//int GROWTH_TIME = 1;
	//std::cout<<"Growth frequency = "<<GROWTH_TIME<<std::endl;
	int translate_frequency = 10;
	std::cout<<"recentering of the model cell frequency = "<<translate_frequency<<std::endl;
	//translate_frequency determines the frequency for the mesh to re-center and perform dynamical remeshing
	int NUMBER_OF_GROWTH_EVENT = 2000;//1000;//1000*2;
	// std::cout<<"Number of maximally allowed growth event = "<<NUMBER_OF_GROWTH_EVENT<<" which used to terminate the simulation if not enough growth is encountered for a prolonged simulation."<<std::endl;
	int NUMBER_OF_TARGETED_GROWTH_EVENT = 1;
	int NKBT = GROWTH_FREQUENCY*NUMBER_OF_GROWTH_EVENT*2;//GROWTH_FREQUENCY*NUMBER_OF_GROWTH_EVENT;//10000;//7500;
	std::cout<<"Number of edge-swap per kBT value (or total number of edge-swap if kBT is fixed), NKBT = "<<NKBT<<std::endl;
	int GROWTH_FREQUENCY_SCALE = 4;
	std::cout<<"GROWTH FREQ SCALE: decides how many growth event must be checked before recording the result = "<<GROWTH_FREQUENCY_SCALE<<std::endl;
	double min_kT = -0.1;//0.21;
	std::cout<<"min kT for simulation termination = "<<min_kT<<std::endl;
	int WHEN = 0;
	
	std::cout<<"Total number of simulation steps: "<<NKBT*max_runTimePerRelaxation<<std::endl;
	double old_total_energy = 0.0;
	double new_total_energy = 0.0;
	double energy_gradient = 0.0;
	double energy_rep = 0.0;
	int Num_of_step_run = 0;
	// auto build_ptr = weak_bld_ptr.lock();//upgrade weak builder to access host variables.
	//std::cout<<"initial LJ-x : "<< ljInfoVecs.LJ_PosX <<std::endl;
	//std::cout<<"initial LJ-y : "<< ljInfoVecs.LJ_PosY <<std::endl;
	//std::cout<<"initial LJ-z : "<< ljInfoVecs.LJ_PosZ <<std::endl;
		

    
	double min_energy;
	generalParams.true_num_edges = 0;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
			generalParams.true_num_edges += 1;
		}
	}
	
	//double COMPRESS = 2.0227;
	// double COMPRESS2 = -2.0227;

	/////////////////////////////////////////////////////////////////
	/////////////////////// MEMBRANE RELATED ////////////////////////
	/////////////////////////////////////////////////////////////////
	
	std::vector<double> nodenormal_1(generalParams.maxNodeCount, 0.0);
	std::vector<double> nodenormal_2(generalParams.maxNodeCount, 0.0);
	std::vector<double> nodenormal_3(generalParams.maxNodeCount, 0.0);
	int reduce_counter = 0;

	double VOLUME_FACTOR = MAX_VOLUME_RATIO;//1.6;//2.25;
	//VOLUME_FACTOR determines the target volume which equals to VOLUME_FACTOR*initial_volume.
	//double tip_depth = 0.5;
	//tip_depth is currently unused.

	double LINE_TENSION_THRESHOLD = -10000.0;
	std::cout<<"LINE TENSION THRESHOLD for activation of line tension = "<<LINE_TENSION_THRESHOLD<<std::endl;
	double VOLUME_THRESHOLD = 0.0;
	std::cout<<"VOLUME THRESHOLD for activation of weakened membrane = "<<VOLUME_THRESHOLD<<std::endl;
	
	double weakened = 1.90;//6.0;
	//weakened determines the minimum height of the z-coordinate of the membrane node to be considered in the area of weakened mechanical properties.
	//double tip_base = 6.0;
	//tip_base currently unused.

	// double EXPAN_THRESHOLD = 0.0;
	// double EXPAN_THRESHOLD_weak = 0.0;//1.75;
	// std::cout<<"EXPANSION THRESHOLD = "<<EXPAN_THRESHOLD<<std::endl;
	// int RULES_OF_EXPAN = 1;	//EXPAN_THRESHOLD is the yielding ratio where a pair of triangles will perform expansion.
	
	// std::cout<<"EXPANSION RULE = "<<RULES_OF_EXPAN<<std::endl;
	// //EXPAN_THRESHOLD_weak is the secondary yielding ratio.
	// //RULES_OF_EXPAN controls how the EXPAN_THRESHOLD is applied:
	// // 1:= Both trianglular areas must exceed the threshold value.
	// // 2:= If one trianglular area exceeds the treshold value while the other exceeds the secondary threshold value.
	// // 3:= If the combined area of the two triangles exceed 2*EXPAN_THRESHOLD.
	// // 4:= If a selected edges exceed the threshold value, split the triangles associated with the edge.

	for (int i = 0; i < generalParams.maxNodeCount; i++){
		generalParams.centerX += coordInfoVecs.nodeLocX[i];
		generalParams.centerY += coordInfoVecs.nodeLocY[i];
		generalParams.centerZ += coordInfoVecs.nodeLocZ[i];
	}
	generalParams.centerX = generalParams.centerX/generalParams.maxNodeCount;
	generalParams.centerY = generalParams.centerY/generalParams.maxNodeCount;
	generalParams.centerZ = generalParams.centerZ/generalParams.maxNodeCount;
	double displacementX, displacementY, displacementZ;
	double newcenterX, newcenterY, newcenterZ;
	//centerX, centerY, centerZ is determined as the referenced origin for recentering of the mesh.

	std::vector<int> VectorShuffleForGrowthLoop;
	std::vector<int> VectorShuffleForFilamentLoop;
	std::vector<int> VectorShuffleForEdgeswapLoop;

	double max_height = coordInfoVecs.nodeLocZ[35];
	double min_height = coordInfoVecs.nodeLocZ[38];
	int max_height_index = 35;
	/*double max_height = -10000.0;
	int max_height_index = -1;
	std::vector<int> Stiffness_gradient();
    for (int k = 0; k < generalParams.maxNodeCount; k++){
        if (coordInfoVecs. nodeLocZ[k] >= max_height){
			max_height = coordInfoVecs. nodeLocZ[k];
			max_height_index = k;
            }
	}*/
	//Max and min height of the membrane nodes, these have to be changed if the mesh used is changed.

	generalParams.Rmin = 0.3012;//0.15;
	//Equilibrium length of an edge of the triangle.
	//generalParams.Rmin_growth = 0.329;
	generalParams.abs_Rmin = generalParams.Rmin;//0.15;
	std::cout<<"abs_Rmin = "<<generalParams.abs_Rmin<<std::endl;
	//Equilibrium distance between membrane node for volume exclusion.
	areaTriangleInfoVecs.initial_area = 0.039;//2835;//0.009808;//0.039;//0.03927344;//0.009817;
	std::cout<<"equilibrium triangular area = "<<areaTriangleInfoVecs.initial_area<<std::endl;
	//Equilibrium triangular area.
	ljInfoVecs.Rmin_M = 0.0;
	//Equilibrium distance between the nucleus particle and membrane.
	ljInfoVecs.Rcutoff_M = 0.0;
	//Maximal interaction range between the nucleus and membrane.
	ljInfoVecs.Rmin_LJ = 0.0;//3.0//1.0;
	//Equilibrium distance between nuclei.
	ljInfoVecs.Rcutoff_LJ = 0.0;//3.0;//1.0;
	//Maximal interaction range between the nuclei.
	ljInfoVecs.epsilon_M_att1 = 0.0;//6.0;//16.0;
	ljInfoVecs.epsilon_M_att2 = 0.0;//1.0;//1.0;
	std::cout<<"Morse_NM_D_att = "<<ljInfoVecs.epsilon_M_att1<<std::endl;
	std::cout<<"Morse_NM_a_att = "<<ljInfoVecs.epsilon_M_att2<<std::endl;
	//Coefficient for the attractive interaction between nuclei and membrane.
	ljInfoVecs.epsilon_M_rep1 = 0.0;//12.5;//16.0;
	ljInfoVecs.epsilon_M_rep2 = 0.0;//0.5;//1.0;
	std::cout<<"Morse_NM_D_rep = "<<ljInfoVecs.epsilon_M_rep1<<std::endl;
	std::cout<<"Morse_NM_a_rep = "<<ljInfoVecs.epsilon_M_rep2<<std::endl;
	//Coefficient for the repulsive interaction between nuclei and membrane.
	
	ljInfoVecs.epsilon_LJ_rep1 = 0.0;//10.0;//0.5;// 0.06;//7.5;
	ljInfoVecs.epsilon_LJ_rep2 = 0.0;//0.5;//1.0;//1.0;//1.0;
	std::cout<<"Morse_NN_D = "<<ljInfoVecs.epsilon_LJ_rep1<<std::endl;
	std::cout<<"Morse_NN_a = "<<ljInfoVecs.epsilon_LJ_rep2<<std::endl;
	//Coefficient of the interaction between nuclei.

	linearSpringInfoVecs.spring_constant_rep1 = 0.01;//0.023;
	linearSpringInfoVecs.spring_constant_rep2 = 9.0;//5.0;
	std::cout<<"Membrane volume exclusion Morse D = "<<linearSpringInfoVecs.spring_constant_rep1<<std::endl;
	std::cout<<"Membrane volume exclusion Morse a = "<<linearSpringInfoVecs.spring_constant_rep2<<std::endl;
	//The coefficient used for non-neighboring membrane node volume exclusion.
	//rep1 is the "D" and rep2 is the "alpha" in the standard form of Morse potential.

	generalParams.volume_spring_constant = 0.2;//(1.0/3.0)*areaTriangleInfoVecs.initial_area*1.0;
	std::cout<<"spring constant for surface normal expansion (pressure within the cell) = "<<generalParams.volume_spring_constant<<std::endl;
	generalParams.line_tension_constant = 0.0;//250.0;
	std::cout<<"spring constant for the septin ring (before budding) = "<<generalParams.line_tension_constant<<std::endl;
	generalParams.length_scale = 1.0;//0.85;//0.1577;//1.0*generalParams.Rmin;// 0.8333;
	//std::cout<<"equilibrium length of each segment of the septin ring = "<<generalParams.length_scale<<std::endl;

	// bendingTriangleInfoVecs.spring_constant = bendingTriangleInfoVecs.spring_constant*(2.0/sqrt(3));

	double scale_linear = linearSpringInfoVecs.spring_constant*1.0;//0.25;//25.0/2.5;//75.0/15.0;
	double scale_bend = bendingTriangleInfoVecs.spring_constant*1.0;//0.05;//10.0/1.0;//75.0/7.5;
	double scale_area = areaTriangleInfoVecs.spring_constant*1.0;//0.25;//50.0/5.0;//75.0/15.0;
	std::cout<<"weakened region linear (before budding) = "<<scale_linear<<std::endl;
	std::cout<<"weakened region bend (before budding) = "<<scale_bend<<std::endl;
	std::cout<<"weakened region area (before budding) = "<<scale_area<<std::endl;
	//linearSpringInfoVecs.spring_constant_weak = linearSpringInfoVecs.spring_constant/scale_linear;
	//bendingTriangleInfoVecs.spring_constant_weak = bendingTriangleInfoVecs.spring_constant/scale_bend;
	//areaTriangleInfoVecs.spring_constant_weak = areaTriangleInfoVecs.spring_constant/scale_area;
	linearSpringInfoVecs.spring_constant_weak = scale_linear;
	bendingTriangleInfoVecs.spring_constant_weak = scale_bend;
	areaTriangleInfoVecs.spring_constant_weak = scale_area;
	//Scaling of the weakend mechanical properties.

	bendingTriangleInfoVecs.initial_angle = 0.087165870975460;//0.087249;//0.04335;
	bendingTriangleInfoVecs.initial_angle_raft = 0.087165870975460;//0.087249;//0.04335;
	bendingTriangleInfoVecs.initial_angle_coat = 0.087165870975460;//0.087249;//0.04335;
	std::cout<<"equilibrium bending angle of the membrane = "<<bendingTriangleInfoVecs.initial_angle<<std::endl;
	//raft and coat are current unused due to the assumption of uniform preferred curvature.

	bendingTriangleInfoVecs.initial_angle_bud = bendingTriangleInfoVecs.initial_angle;
	std::cout<<"equilibrium bending angle of the bud = "<<bendingTriangleInfoVecs.initial_angle_bud<<std::endl;
	
	// bendingTriangleInfoVecs.spring_constant_raft = 0.0;//bendingTriangleInfoVecs.spring_constant;
	// bendingTriangleInfoVecs.spring_constant_coat = 0.0;//bendingTriangleInfoVecs.spring_constant;
	// bendingTriangleInfoVecs.spring_constant = bendingTriangleInfoVecs.spring_constant*(2.0/sqrt(3));
	// bendingTriangleInfoVecs.spring_constant_raft = bendingTriangleInfoVecs.spring_constant_raft*(2.0/sqrt(3));
	// bendingTriangleInfoVecs.spring_constant_coat = bendingTriangleInfoVecs.spring_constant_coat*(2.0/sqrt(3));
	// std::cout<<"Effective bending coefficient is calculated by multiplying 2/sqrt(3)"<<std::endl;
	// std::cout<<"effective bending coefficient of the membrane = "<<bendingTriangleInfoVecs.spring_constant<<std::endl;
	// std::cout<<"effective bending coefficient of the membrane raft = "<<bendingTriangleInfoVecs.spring_constant_raft<<std::endl;
	// std::cout<<"effective bending coefficient of the membrane coat = "<<bendingTriangleInfoVecs.spring_constant_coat<<std::endl;

	std::vector<int> pull_nodes_up;// = {35,    76,    79,   111,   113,   151,   153,   360,   361,   362,   363,   364,   365,   505,   506,   515,   516,   593,   632};//{35, 360,   361,   362,   363,   364,   365};
	std::vector<int> pull_nodes_down;// = {38,    86,    89,   121,   123,   144,   146,   378,   379,   380,   381,   382,   383,   535,   536,   545,   546,   602,   626};//{38, 378,   379,   380,   381,   382,   383};
	std::vector<int> push_nodes_down;
	std::vector<int> push_nodes_up;
	// for (int i = 0; i < generalParams.maxNodeCount; i++){
	// 	if (coordInfoVecs.nodeLocZ[i] >= 1.43026488631){
	// 		pull_nodes_up.push_back(i);
	// 	}
	// 	if (coordInfoVecs.nodeLocZ[i] <= -1.43026488631){
	// 		pull_nodes_down.push_back(i);
	// 	}
	// }

	/////////////////////////////////////////////////////////////////
	////////////////// END OF MEMBRANE RELATED //////////////////////
	/////////////////////////////////////////////////////////////////


	/////////////////////////////////////////////////////////////////
	//////////////////////// NULCEUS RELATED ////////////////////////
	/////////////////////////////////////////////////////////////////
	double beta1 = 0.0;
	double beta2 = 0.0;
	std::cout<<"manual push speed for the nucleus tip = "<<beta1<<std::endl;
	std::cout<<"manual push speed for the remainder of the nucleus = "<<beta2<<std::endl;
	//beta1 is the vertical speed (0, 0, beta1) applied to the nucleus tip.
	//beta2 is the vertical speed (0, 0, beta2) applied to the remainder of the nucleus.

	std::vector<double> V1 = {-0.0};/*, 0.0  ,  0.1966  ,  0.5547 ,  -0.4689 ,   0.2422 ,  -0.2229,
							   -0.4312 ,  -0.0185 ,   0.2887 ,   0.3187 ,   0.7140 ,  
								0.2231 ,  -0.1921 ,	  -0.5541 ,   -0.1542 ,   -0.1689 ,    0.4391 ,
							   -0.6661 ,  -0.6381 ,   0.6256 ,   0.0466 ,  -0.0610 ,   0.5134};
								*/
	std::vector<double> V2 = {0.0};/*, 0.0 ,  -0.4595 ,  -0.4129 ,   0.0954 ,   0.1764 ,   0.4186 ,
							  -0.5602 ,  -0.6082 ,  -0.5318 ,   0.3561 ,   0.0753 ,
							  -0.0917 ,  -0.2596 , 0.2871 ,  -0.3918 ,   0.5195 ,   0.5579 ,
							  -0.2805 ,   0.0133  , -0.0073 ,   0.7426 ,   0.0614 ,  -0.1506};
								*/
	std::vector<double> V3 = { 0.6390};/*, 0.0 ,  -0.5511 ,   0.0267 ,  -0.5240  , -0.4004 ,   0.2850 ,
							   0.2032 ,  -0.1771 ,   0.4048 ,   0.3461 ,  -0.2034 ,
							   0.5041 ,  -0.4535 ,	-0.1241 ,   0.5722 ,  -0.3748 ,  -0.1335 ,
							   -0.0851 ,   0.3213 ,   0.2389 ,   0.0044 ,  -0.7424 ,  -0.7450};
							   */
	//V1, V2, and V3 are the (x,y,z)-coordinate of the nucleus particles.

	for (int i = 0; i < V1.size(); i++){
		ljInfoVecs.LJ_PosX_all.push_back(V1[i]); 
		ljInfoVecs.LJ_PosY_all.push_back(V2[i]);
		ljInfoVecs.LJ_PosZ_all.push_back(V3[i]);
	}  
	
	double NUCLEUS_UPPERHEM_BASE = 0.5;
	double NUCLEUS_LOWERHEM_BASE = -0.6;
	//These values defines the z-coordinate requirement for nucleus particles to be considered tip-region or base-region. This is used to 
	// determine where to apply spring or constant force.

	//////////////////////////////////////////////////////////////////
	///////////////// END OF NUCLEUS RELATED /////////////////////////
	//////////////////////////////////////////////////////////////////

	/*std::vector<int> filament_base(generalParams.maxNodeCountLJ, -1); //= {0,1,2,3,4,5,6,7,8,9,10,11};//{35, 21, 38, etc if we need more points}
	double filament_strength = 0.0;
	double filament_strength_pull = 1.0*filament_strength;
	double filament_Rmin = ((max_height - min_height)/4.0);
	std::cout<<"filament_strength = "<<filament_strength<<std::endl;
	std::cout<<"filament_strength for vertical pull = "<<filament_strength_pull<<std::endl;
	std::cout<<"filament_Rmin = "<<filament_Rmin<<std::endl;
	
	//First, determine the initial membrane nodes having filament bridges
	//with the nuclei particles
	for (int i = 0; i < generalParams.maxNodeCountLJ; i++){
		if (i == 0){
			filament_base[i] = 35;
			continue;
		}
		for (int j = 0; j < generalParams.maxNodeCount; j++){
			double xsquared = (ljInfoVecs.LJ_PosX_all[i] - coordInfoVecs.nodeLocX[j])*
								(ljInfoVecs.LJ_PosX_all[i] - coordInfoVecs.nodeLocX[j]);
			double ysquared = (ljInfoVecs.LJ_PosY_all[i] - coordInfoVecs.nodeLocY[j])*
								(ljInfoVecs.LJ_PosY_all[i] - coordInfoVecs.nodeLocY[j]);
			double zsquared = (ljInfoVecs.LJ_PosZ_all[i] - coordInfoVecs.nodeLocZ[j])*
								(ljInfoVecs.LJ_PosZ_all[i] - coordInfoVecs.nodeLocZ[j]);
			double R = sqrt(xsquared + ysquared + zsquared);
			if (R < filament_Rmin*1.1 && j != 35){
				filament_base[i] = j;
				break;
			}
		}
	}*/
	
	//std::vector<double> filament_Rmin;
	//for (int i = 0; i < V3.size();i++){
	//	filament_Rmin.push_back(sqrt((V3[i] - coordInfoVecs.nodeLocZ[38])*(V3[i] - coordInfoVecs.nodeLocZ[38])));
	//}
	//double filament_Rmin = sqrt((V3.back() - coordInfoVecs.nodeLocZ[38])*(V3.back() - coordInfoVecs.nodeLocZ[38]));
	//This part calculates the filament connecting the minimum point (in terms of z-coordinate) to the base of the nuclei cluster.


	//////////////////////////////////////////////////////////////////
	/////////// IDENTIFYING REGIONS WITH DIFFERENT MECH PROP /////////
	//////////////////////////////////////////////////////////////////

	/*ljInfoVecs.forceX_all.reserve(ljInfoVecs.LJ_PosX_all.size());
	ljInfoVecs.forceY_all.reserve(ljInfoVecs.LJ_PosX_all.size());
	ljInfoVecs.forceZ_all.reserve(ljInfoVecs.LJ_PosX_all.size());

	generalParams.maxNodeCountLJ = ljInfoVecs.LJ_PosX_all.size();
	std::vector<int> nucleus_in_upperhem(generalParams.maxNodeCountLJ, -1);
	std::vector<int> nucleus_in_lowerhem(generalParams.maxNodeCountLJ, -1);
	for (int i = 0; i < generalParams.maxNodeCountLJ; i++){
		if (ljInfoVecs.LJ_PosZ_all[i] > NUCLEUS_UPPERHEM_BASE){
			nucleus_in_upperhem[i] = 1;
		}
		if (ljInfoVecs.LJ_PosZ_all[i] < NUCLEUS_LOWERHEM_BASE){
			nucleus_in_lowerhem[i] = 1;
		}
	}*/
	

	std::vector<int> out;
	//int ALPHA;

	std::vector<bool> boundary_edges;
	boundary_edges.reserve(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		if (coordInfoVecs.edges2Triangles_1[i] == coordInfoVecs.edges2Triangles_2[i]){
			boundary_edges.push_back(true);
		}
		else {
			boundary_edges.push_back(false);
		}
	}

	std::vector<int> edgeIndices;
	edgeIndices.reserve(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; ++i){
		//edgeIndices.push_back(edge_to_ljparticle[i]);
		if (boundary_edges[i] == false){
			edgeIndices.push_back(i);
		}
		else {
			edgeIndices.push_back(-1);
		}
	}



	auto it = remove_if(edgeIndices.begin(), edgeIndices.end(),  [](const int i) {return i < 0; });
	edgeIndices.erase(it, edgeIndices.end());
	
	std::vector<int> row2 = {35 ,   76 ,   79 ,  111 ,  113 ,  151 ,  153 ,  360 ,  361 ,  362 ,  363 ,  364 ,  365 ,  505 ,  506 ,  515 ,  516 ,  593 ,  632};
	// std::vector<int> row2 = {35,76,79,111,113,151,153,360,361,362,363,364,365,505,506,515,516,593,632,840,841,842,
	//    843,844,845,1087,1090,1091,1105,1108,1109,1297,1299,1301,1309,1311,1313,1537,1539,1541,1549,1551,1553,2196,
	//   2197,2198,2199,2200,2201,2202,2203, 2204, 2205,2206,2207, 2208,2209,2210,2211,2212,2213};
	//std::vector<int> nodes_to_center;
	//generalParams.nodes_in_upperhem.resize(generalParams.maxNodeCount,-1);

	for (int i = 0; i < generalParams.maxNodeCount; i++){
		generalParams.nodes_in_upperhem[i] = -1;
		// generalParams.nodes_in_upperhem[i] = 1;
	}

	for (int i = 0; i < row2.size(); i++){
		generalParams.nodes_in_upperhem[row2[i]] = 1;
	//	std::cout<<"nodes "<<i<<" "<<generalParams.nodes_in_upperhem[i]<<std::endl;		
	}
	// for (int i = 0; i < generalParams.maxNodeCount; i++){
	// 	if (coordInfoVecs.nodeLocZ[i] > (generalParams.centerZ + weakened)){
	// 		generalParams.nodes_in_upperhem[i] = 1;
	// 	}
	// 	else{
	// 		generalParams.nodes_in_upperhem[i] = -1;
	// 	}
	// //	std::cout<<"nodes "<<i<<" "<<generalParams.nodes_in_upperhem[i]<<std::endl;		
	// }

	//std::vector<int> nodes_to_center;
	//std::vector<int> nodes_in_tip;
	//nodes_in_tip.resize(generalParams.maxNodeCount);
	//for (int i = 0; i < generalParams.maxNodeCount; i++){
	//	if (coordInfoVecs.nodeLocZ[i] > (generalParams.centerZ + tip_base)){
	//		nodes_in_tip[i] = 1;
	//	}
	//	else{
	//		nodes_in_tip[i] = -1;
	//	}
	//	std::cout<<"nodes "<<i<<" "<<generalParams.nodes_in_upperhem[i]<<std::endl;		
	//}

	//generalParams.triangles_in_upperhem.resize(coordInfoVecs.num_triangles);
	for (int i = 0; i < coordInfoVecs.num_triangles; i++){
		if (coordInfoVecs.triangles2Nodes_1[i] >= (INT_MAX-1000) || coordInfoVecs.triangles2Nodes_1[i] < 0){
			generalParams.triangles_in_upperhem[i] = -1;
			continue;
		}
		else if (coordInfoVecs.triangles2Nodes_2[i] >= (INT_MAX-1000) || coordInfoVecs.triangles2Nodes_2[i] < 0){
			generalParams.triangles_in_upperhem[i] = -1;
			continue;
		}
		else if (coordInfoVecs.triangles2Nodes_3[i] >= (INT_MAX-1000) || coordInfoVecs.triangles2Nodes_3[i] < 0){
			generalParams.triangles_in_upperhem[i] = -1;
			continue;
		}
		int aaa = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_1[i]];
		//std::cout<<aaa<<std::endl;
		int bbb = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_2[i]];
		//std::cout<<bbb<<std::endl;
		int ccc = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_3[i]];
		//std::cout<<ccc<<std::endl;
		if ((aaa+bbb+ccc)==3){
			generalParams.triangles_in_upperhem[i] = 1;
			//triangles_in_upperhem.push_back(i);
		}
		//else if ((aaa+bbb+ccc)==1){
		//	generalParams.triangles_in_upperhem[i] = 0;
			//triangles_in_upperhem.push_back(i);
		//}
		else{
			generalParams.triangles_in_upperhem[i] = -1;
		}
	//	std::cout<<"triangle "<<i<<" "<<generalParams.triangles_in_upperhem[i]<<std::endl;		
	}

	//std::vector<int> edges_in_upperhem;
//	generalParams.edges_in_upperhem.resize(coordInfoVecs.num_edges);
	int edges_in_upperhem_COUNT = 0;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		if (coordInfoVecs.edges2Triangles_1[i] >= (INT_MAX-1000) || coordInfoVecs.edges2Triangles_1[i] < 0){
			generalParams.edges_in_upperhem[i] = -1;
			generalParams.edges_in_upperhem_list[i] = -INT_MAX;
			// std::cout<<"Filter1, edge = "<<i<<std::endl;
			continue;
		}
		else if (coordInfoVecs.edges2Triangles_2[i] >= (INT_MAX-1000) || coordInfoVecs.edges2Triangles_2[i] < 0){
			generalParams.edges_in_upperhem[i] = -1;
			generalParams.edges_in_upperhem_list[i] = -INT_MAX;
			// std::cout<<"Filter2, edge = "<<i<<std::endl;
			continue;
		}
		else{
			// std::cout<<"Filter 3, edge = "<<i<<std::endl;
			int aaa = generalParams.triangles_in_upperhem[coordInfoVecs.edges2Triangles_1[i]];//generalParams.nodes_in_upperhem[coordInfoVecs.edges2Nodes_1[i]];
			int bbb = generalParams.triangles_in_upperhem[coordInfoVecs.edges2Triangles_2[i]];//generalParams.nodes_in_upperhem[coordInfoVecs.edges2Nodes_2[i]];
			if (aaa == 1 && bbb == 1){
				generalParams.edges_in_upperhem[i] = 1;
				//generalParams.edges_in_upperhem_list.push_back(i);
				generalParams.edges_in_upperhem_list[i] = i;
				edges_in_upperhem_COUNT += 1;
			}
			else if (aaa == 1 || bbb == 1){
				generalParams.edges_in_upperhem[i] = 1;
				generalParams.edges_in_upperhem_list[i] = -INT_MAX;
				edges_in_upperhem_COUNT += 1;
			}
			else{
				generalParams.edges_in_upperhem[i] = -1;
				generalParams.edges_in_upperhem_list[i] = -INT_MAX;
			}
		}
		// std::cout<<edges_in_upperhem_COUNT<<std::endl;
		
	}
	std::cout<<"INITIAL EDGES IN UPPERHEM = "<<edges_in_upperhem_COUNT<<std::endl;

	int COUNTING_EDGE = 0;
	for (int y = 0; y < coordInfoVecs.num_edges; y++){
		if (generalParams.edges_in_upperhem_list[y] >= 0){
			COUNTING_EDGE += 1;
		}
		generalParams.edges_in_upperhem_list_length = COUNTING_EDGE;
	}
	

	//Find the boundary of the nodes_in_upperhem region
	//generalParams.boundaries_in_upperhem.resize(coordInfoVecs.num_edges);
	std::vector<int> boundary_node_list;
	std::vector<int> boundary_edge_list;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		double T1 = coordInfoVecs.edges2Triangles_1[i];
		double T2 = coordInfoVecs.edges2Triangles_2[i];
		if (T1 >= (INT_MAX - 1000) || T1 < 0 || T2 >= (INT_MAX-1000) || T2 < 0){
			continue;
		}
		if (generalParams.triangles_in_upperhem[T1] == 1 && generalParams.triangles_in_upperhem[T2] != 1){
			generalParams.boundaries_in_upperhem[i] = 1;
			//std::cout<<generalParams.boundaries_in_upperhem[i]<<std::endl;
		//	generalParams.triangles_in_upperhem[T1] = 0;
		//	generalParams.triangles_in_upperhem[T2] = 0;
			double bdry_node1 = coordInfoVecs.edges2Nodes_1[i];
			double bdry_node2 = coordInfoVecs.edges2Nodes_2[i];
			boundary_node_list.push_back(bdry_node1);
			boundary_node_list.push_back(bdry_node2);
			boundary_edge_list.push_back(i);
			//generalParams.nodes_in_upperhem[bdry_node1] = 0;
			//generalParams.nodes_in_upperhem[bdry_node2] = 0;
			// coordInfoVecs.isNodeFixed[bdry_node1] = true;
			// coordInfoVecs.isNodeFixed[bdry_node2] = true;
		}
		else if (generalParams.triangles_in_upperhem[T1] != 1 && generalParams.triangles_in_upperhem[T2] == 1){
			generalParams.boundaries_in_upperhem[i] = 1;
			//std::cout<<generalParams.boundaries_in_upperhem[i]<<std::endl;
		//	generalParams.triangles_in_upperhem[T1] = 0;
		//	generalParams.triangles_in_upperhem[T2] = 0;
			double bdry_node1 = coordInfoVecs.edges2Nodes_1[i];
			double bdry_node2 = coordInfoVecs.edges2Nodes_2[i];
			boundary_node_list.push_back(bdry_node1);
			boundary_node_list.push_back(bdry_node2);
			boundary_edge_list.push_back(i);
			//generalParams.nodes_in_upperhem[bdry_node1] = 0;
			//generalParams.nodes_in_upperhem[bdry_node2] = 0;
			// coordInfoVecs.isNodeFixed[bdry_node1] = true;
			// coordInfoVecs.isNodeFixed[bdry_node2] = true;
		}
		else {
			generalParams.boundaries_in_upperhem[i] = -1;
			//std::cout<<generalParams.boundaries_in_upperhem[i]<<std::endl;
		}
	}
	std::cout<<"size of boundary_node_list (this is double-counted) = "<<boundary_node_list.size()<<std::endl;
	//generalParams.eq_total_boundary_length = generalParams.boundaries_in_upperhem.size()*generalParams.Rmin;

	/*for (int i = 0; i < coordInfoVecs.num_edges; i++){
		int aaa = coordInfoVecs.edges2Nodes_1[i];
		int bbb = coordInfoVecs.edges2Nodes_2[i];
		if (aaa == 1 && bbb == 1){
			generalParams.edges_in_upperhem[i] = 1;
			generalParams.edges_in_upperhem_list.push_back(i);
		}
		else if (aaa == 1 || bbb == 1){
			generalParams.edges_in_upperhem[i] = 0;
		}
		else{
			generalParams.edges_in_upperhem[i] = -1;
		}
		
	}*/
	
	

	int true_num_edges_in_upperhem = 0;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		if (generalParams.edges_in_upperhem_list[i] != INT_MAX && generalParams.edges_in_upperhem_list[i] >= 0){
		true_num_edges_in_upperhem += 1;
		}
	}
	

	//std::vector<int> edge_to_ljparticle;
	//generalParams.edge_to_ljparticle.reserve(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		generalParams.edge_to_ljparticle.push_back(-1);
	};
	/////////////////////////////////////////////////////////////////////
	////////////// END OF IDENTIFYING REG. WITH DIFF. MECH PROP /////////
	/////////////////////////////////////////////////////////////////////


	//std::cout<<"ERROR HERE?"<<std::endl;
	ComputeVolume(
		generalParams,
		coordInfoVecs,
		linearSpringInfoVecs,
		ljInfoVecs
	);
	//std::cout<<"ERROR HERE 2?"<<std::endl;
	double initial_volume;
	// initial_volume = generalParams.true_current_total_volume;
	// generalParams.eq_total_volume = generalParams.true_current_total_volume*VOLUME_FACTOR;//This is for setting different equilibrium volume to mimic growth or shirnkage.
	// std::cout<<"true current total volume = "<<generalParams.true_current_total_volume<<std::endl;
	// std::cout<<"eq total volume = "<<generalParams.eq_total_volume<<std::endl;

	//////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////// START OF ACTUAL SIMULATION /////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////////

	/* Build the initial gradient weakend scale */
	dtb = 0.0;//dtb := distance to boundary
	generalParams.septin_ring_z = 0.0;
	generalParams.boundary_z = 0.0;
	//for (int k = 0; k < boundary_edge_list.size(); k++){
	for (int k = 0; k < boundary_node_list.size(); k++){
		double n1 = boundary_node_list[k];//coordInfoVecs.edges2Nodes_1[boundary_edge_list[k]];
		//double n2 = coordInfoVecs.edges2Nodes_2[boundary_edge_list[k]];
		//double cent_of_edge_x = (coordInfoVecs.nodeLocX[n1] + coordInfoVecs.nodeLocX[n2])/2.0;
		//double cent_of_edge_y = (coordInfoVecs.nodeLocY[n1] + coordInfoVecs.nodeLocY[n2])/2.0;
		//double cent_of_edge_z = (coordInfoVecs.nodeLocZ[n1] + coordInfoVecs.nodeLocZ[n2])/2.0;
		double dist_x = coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1];//cent_of_edge_x;
		double dist_y = coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1];//cent_of_edge_y;
		double dist_z = coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1];//cent_of_edge_z;
		// double temp_dist = sqrt((coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1])*(coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1]) +
		// (coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1])*(coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1]) +
		// 	(coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1])*(coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1]));
		double temp_dist = sqrt(dist_x*dist_x + dist_y*dist_y + dist_z*dist_z);
		generalParams.septin_ring_z += coordInfoVecs.nodeLocZ[n1];
		if (temp_dist >= dtb){
			dtb = temp_dist;
			/* "dtb" will be used to identify where the septin ring is located, and used to determine the Hill coefficient*/
		}
	}
	std::cout<<"dtb = "<<dtb<<std::endl;
	//generalParams.septin_ring_z = generalParams.septin_ring_z/boundary_node_list.size();
	//generalParams.boundary_z = generalParams.septin_ring_z - generalParams.Rmin;
	/* dtb will be only calculated once so we can effectively keep the Hill eqn curve consistent with only horizontal shift */
	dtb_max = dtb + (generalParams.Rmin);
	
	std::cout<<"initial distance between cell tip and the boundary of weakened area = "<<dtb<<std::endl;
	std::cout<<"Notice that here, the distance from the tip to the boundary is slightly extended by half of the equilibrium length of an edge"<<std::endl;
	//std::cout<<"If this message is present, we are forcing a fixed portion of the bud tip to be occupied by the max concentration"<<std::endl;
	//generalParams.hilleqnconst = (dtb + generalParams.Rmin/4.0)/dtb_max;
	generalParams.hilleqnconst = dtb/dtb_max;
	generalParams.hilleqnpow = 70.0;
	// std::cout<<"hill equation constant K = "<<generalParams.hilleqnconst<<std::endl;
	// std::cout<<"hill (equation) coefficient = "<<generalParams.hilleqnpow<<std::endl;
	// std::cout<<"NOTE: IN THIS SIMULATION, THE LOCATION WHERE 50% WEAKENING IS EXPERIENCED IS LOCATED SLIGHTLY AWAY FROM THE SEPTIN RING, "<<std::endl;
	// std::cout<<"THIS IS DUE TO THE FACT THAT IN ISOTROPIC CASE, SEPTIN RING LOCATION MUST BE SUFFICIENTLY WEAKENED TO INDUCE BUDDING"<<std::endl;
	// std::cout<<" "<<std::endl;
	// std::cout<<" "<<std::endl;
	// std::cout<<" "<<std::endl;
	// std::cout<<" "<<std::endl;
	// std::cout<<" "<<std::endl;
	// std::cout<<" "<<std::endl;
	// std::cout<<" "<<std::endl;
	// std::cout<<" "<<std::endl;


	utilities_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
	std::cout<<"HA?"<<std::endl;
	utilities_ptr->gradient_weakening_update_host_vecs(sigma,
		//max_height_index,
		coordInfoVecs.nodeLocX[max_height_index],
		coordInfoVecs.nodeLocY[max_height_index],
		coordInfoVecs.nodeLocZ[max_height_index],
		dtb,
		dtb_max,
		generalParams,
		coordInfoVecs,
		build_ptr->hostSetInfoVecs);
		std::cout<<"HAHA?"<<std::endl;
	for (int u = 0; u < generalParams.maxNodeCount; u++){
		// std::cout<<"u = "<<u<<std::endl;
		int BETA = utilities_ptr->nodes2Triangles_host_vecs(
			u,
			build_ptr->hostSetInfoVecs,
			coordInfoVecs,
			generalParams,
			auxVecs);
	}
	utilities_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
	std::cout<<"STARTING THE ACTUAL SIMULATION"<<std::endl;
	while (runSim == true){
		
		double current_time = 0.0;

		int translate_counter = 0;
		
        while (current_time < relax_max_steps_before_growth_and_edgeswap*(Max_Runtime)){
            translate_counter += 1;
            Solve_Forces();
            double beta;
                
            AdvancePositions(
                coordInfoVecs,
                generalParams,
                domainParams);
                        
            new_total_energy = linearSpringInfoVecs.linear_spring_energy + 
                areaTriangleInfoVecs.area_triangle_energy + 
                bendingTriangleInfoVecs.bending_triangle_energy;// + 
                0.5*energy_rep;// + 
                //ljInfoVecs.lj_energy_M +
                //ljInfoVecs.lj_energy_LJ +
                //generalParams.volume_energy;

            // energy_gradient = sqrt((new_total_energy - old_total_energy)*(new_total_energy - old_total_energy))/old_total_energy;
            // if (current_time >= Max_Runtime*minimal_run_time_ratio && (energy_gradient/generalParams.dt) < energy_gradient_threshold){
            //     break;
            //     }
            old_total_energy = new_total_energy;
            current_time+=generalParams.dt;
        }

		std::cout<<"Time used for 'steady state' initial condition before growth and edge swaps = "<<current_time<<std::endl;
		std::cout<<"current total energy (before growth and edge swaps) = "<<new_total_energy<<std::endl;
		std::cout<<"LINEAR ENERGY = "<<linearSpringInfoVecs.linear_spring_energy<<std::endl;
		std::cout<<"BEND ENERGY = "<<bendingTriangleInfoVecs.bending_triangle_energy<<std::endl;
		std::cout<<"AREA ENERGY = "<<areaTriangleInfoVecs.area_triangle_energy<<std::endl;
		//std::cout<<"REPULSION ENERGY = "<<energy_rep<<std::endl;
		std::cout<<"VOLUME ENERGY = "<<generalParams.volume_energy<<std::endl;
		std::cout<<"true_current_total_volume (before growth and edge swaps) = "<<generalParams.true_current_total_volume<<std::endl;
		// std::cout<<"eq_total_volume = "<<generalParams.eq_total_volume<<std::endl;
		std::cout<<"current KBT = "<<generalParams.kT<<std::endl;
		if (isnan(new_total_energy)==1){
			std::cout<<"Nan or Inf position update !!!!"<<std::endl;
			runSim = false;
			break;
		}
		double current_bud_area = 0.0;
		for (int k = 0; k < coordInfoVecs.num_triangles; k++){
			if (coordInfoVecs.triangles2Nodes_1[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_1[k] <= (-INT_MAX + 1000.0) ||
				coordInfoVecs.triangles2Nodes_2[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_2[k] <= (-INT_MAX + 1000.0) ||
				coordInfoVecs.triangles2Nodes_3[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_3[k] <= (-INT_MAX + 1000.0)){
						continue;
					}
			else{
				if (generalParams.triangles_in_upperhem[k] == 1){
					double r1x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[k]];
					double r1y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[k]];
					double r1z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[k]];
					double r2x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_2[k]];
					double r2y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_2[k]];
					double r2z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[k]];
					double r3x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_3[k]];
					double r3y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_3[k]];
					double r3z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[k]];
					double norm_r1r2 = sqrt((r2x-r1x)*(r2x-r1x) + (r2y-r1y)*(r2y-r1y) + (r2z-r1z)*(r2z-r1z));
					double norm_r2r3 = sqrt((r3x-r2x)*(r3x-r2x) + (r3y-r2y)*(r3y-r2y) + (r3z-r2z)*(r3z-r2z));
					double norm_r3r1 = sqrt((r3x-r1x)*(r3x-r1x) + (r3y-r1y)*(r3y-r1y) + (r3z-r1z)*(r3z-r1z));
					double s = (norm_r1r2 + norm_r2r3 + norm_r3r1)/2.0;
					double area = sqrt(s*(s-norm_r1r2)*(s-norm_r2r3)*(s-norm_r3r1));
					current_bud_area += area;
				}
			}
		}
		double Initial_Bud_Area = current_bud_area;
		std::cout<<"Initial bud surface area (before growth and edge swaps) = "<<Initial_Bud_Area<<std::endl;

		generalParams.volume_spring_constant = 0.2;//(1.0/3.0)*areaTriangleInfoVecs.initial_area*1.0;
		std::cout<<"spring constant for surface normal expansion (pressure within the cell) = "<<generalParams.volume_spring_constant<<std::endl;
		generalParams.line_tension_constant = 50.0;//250.0;
		std::cout<<"spring constant for the septin ring = "<<generalParams.line_tension_constant<<std::endl;
		generalParams.length_scale = 1.0;//0.85;//0.1577;//1.0*generalParams.Rmin;// 0.8333;
		//std::cout<<"equilibrium length of each segment of the septin ring = "<<generalParams.length_scale<<std::endl;

		double scale_linear = linearSpringInfoVecs.spring_constant*0.75;//0.25;//25.0/2.5;//75.0/15.0;
		double scale_bend = bendingTriangleInfoVecs.spring_constant*0.135;//0.05;//10.0/1.0;//75.0/7.5;
		double scale_area = areaTriangleInfoVecs.spring_constant*0.75;//0.25;//50.0/5.0;//75.0/15.0;
		std::cout<<"weakened region linear = "<<scale_linear<<std::endl;
		std::cout<<"weakened region bend = "<<scale_bend<<std::endl;
		std::cout<<"weakened region area = "<<scale_area<<std::endl;
		//linearSpringInfoVecs.spring_constant_weak = linearSpringInfoVecs.spring_constant/scale_linear;
		//bendingTriangleInfoVecs.spring_constant_weak = bendingTriangleInfoVecs.spring_constant/scale_bend;
		//areaTriangleInfoVecs.spring_constant_weak = areaTriangleInfoVecs.spring_constant/scale_area;
		linearSpringInfoVecs.spring_constant_weak = scale_linear;
		bendingTriangleInfoVecs.spring_constant_weak = scale_bend;
		areaTriangleInfoVecs.spring_constant_weak = scale_area;
		//Scaling of the weakend mechanical properties.
		initial_volume = generalParams.true_current_total_volume;
		generalParams.eq_total_volume = generalParams.true_current_total_volume*VOLUME_FACTOR;//This is for setting different equilibrium volume to mimic growth or shirnkage.
		std::cout<<"true current total volume = "<<generalParams.true_current_total_volume<<std::endl;
		std::cout<<"eq total volume = "<<generalParams.eq_total_volume<<std::endl;
	
		storage->print_VTK_File();
		storage->storeVariables();

		utilities_ptr->LDG_Surface_Diffusion_Initialize(
			coordInfoVecs,
			generalParams,
			build_ptr->hostSetInfoVecs,
			auxVecs,
			rbc,
			n_rbc,
			mem_prealloc
		);

		int edgeswap_iteration = 0;
		num_edge_loop = 0;//round(true_num_edges_in_upperhem*SAMPLE_SIZE);	

		int LINE_TENSION_START = 0;
		
		bool WEAKENED_START = false;
		bool EDGESWAP_ALGORITHM_TRIGGERED;
		bool needToRebuildDiffStructAfterEdgeSwap = false;
		int number_of_simulation_step = 0;
 		while (initial_kT > 0){
			if (edgeswap_iteration >= NKBT){
				runSim = false;
				initial_kT = -1;
				break;
			}
			////////////////////NOW RELAX THE ATTEMPTED EDGESWAP//////////////////////
				current_time = 0.0;
				translate_counter = 0;
				double VOLUME_RATIO = generalParams.true_current_total_volume/generalParams.eq_total_volume;
			
			if (generalParams.true_current_total_volume/initial_volume >= LINE_TENSION_THRESHOLD && edgeswap_iteration == 0){
			// 	if (LINE_TENSION_START < 1){
					double DIST = 0.0;
					double COUNT = 0.0;
					for (int t = 0; t < coordInfoVecs.num_edges; t++){
						if (generalParams.boundaries_in_upperhem[t] == 1){
							COUNT += 1.0;
							int node1 = coordInfoVecs.edges2Nodes_1[t];
							int node2 = coordInfoVecs.edges2Nodes_2[t];
							DIST += sqrt((coordInfoVecs.nodeLocX[node2] - coordInfoVecs.nodeLocX[node1])*(coordInfoVecs.nodeLocX[node2] - coordInfoVecs.nodeLocX[node1]) +
							(coordInfoVecs.nodeLocY[node2] - coordInfoVecs.nodeLocY[node1])*(coordInfoVecs.nodeLocY[node2] - coordInfoVecs.nodeLocY[node1]) + 
							(coordInfoVecs.nodeLocZ[node2] - coordInfoVecs.nodeLocZ[node1])*(coordInfoVecs.nodeLocZ[node2] - coordInfoVecs.nodeLocZ[node1]));
						}
					}
					// for (int t = 0; t < coordInfoVecs.num_edges; t++){
					// 	if (generalParams.boundaries_in_upperhem[t] == 1){
					// 		COUNT += 1.0;
					// 	}
					// }
					generalParams.length_scale = (DIST/COUNT)/generalParams.Rmin;
					std::cout<<"equilibrium length of each segment of the septin ring = "<<generalParams.length_scale*generalParams.Rmin<<std::endl;
					generalParams.eq_total_boundary_length = COUNT*generalParams.length_scale* generalParams.Rmin;
					std::cout<<"equilibrium length of the septin ring = "<<generalParams.eq_total_boundary_length<<std::endl;
					LINE_TENSION_START += 1;
			// 	}
				
			}
			//std::cout<<"start relaxation step"<<std::endl;
			EDGESWAP_ALGORITHM_TRIGGERED = false;
			bool end_of_relaxation = false;
			while (current_time < Max_Runtime){
				number_of_simulation_step += 1;
				if (Max_Runtime <= 0.0){
					std::cout<<"Max_Runtime is set to be 0 or negative! "<<std::endl;
					break;
				}
					
				Solve_Forces();
					
				if (LINE_TENSION_START >= 1){
					ComputeLineTensionSprings(
						generalParams,
						coordInfoVecs,
						linearSpringInfoVecs);
					}
				//std::cout<<"STOPPED BEFORE MemRepul"<<std::endl;
				/*energy_rep =
				ComputeMemRepulsionEnergy(
					coordInfoVecs,
					linearSpringInfoVecs, 
					capsidInfoVecs,
					generalParams,
						auxVecs);*/						

				AdvancePositions(
					coordInfoVecs,
					generalParams,
						domainParams);

				new_total_energy = linearSpringInfoVecs.linear_spring_energy + 
			areaTriangleInfoVecs.area_triangle_energy + 
			bendingTriangleInfoVecs.bending_triangle_energy;// +
			0.5*energy_rep;

		energy_gradient = sqrt((new_total_energy - old_total_energy)*(new_total_energy - old_total_energy))/old_total_energy;
		old_total_energy = new_total_energy;
		current_time+=generalParams.dt;	
	
				if (translate_counter % translate_frequency == 0){
					//	std::cout<<"SIMULATIONs TRIGGER REPOSITIONING AND EDGESWAP?"<<std::endl;

					newcenterX = 0.0;
					newcenterY = 0.0;
					newcenterZ = 0.0;
				//	std::cout<<"HERE?"<<std::endl;
					
					for (int i = 0; i < generalParams.maxNodeCount; i++){//for (int i = 0; i < coordInfoVecs.nodeLocX.size(); i++){
						//std::cout<<i<<std::endl;
						newcenterX += coordInfoVecs.nodeLocX[i];
						//std::cout<<newcenterX<<std::endl;
						newcenterY += coordInfoVecs.nodeLocY[i];
						//std::cout<<newcenterY<<std::endl;
						newcenterZ += coordInfoVecs.nodeLocZ[i];
						//std::cout<<newcenterZ<<std::endl;
					}
				//	std::cout<<"HERE2?"<<std::endl;
					newcenterX = newcenterX/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
					newcenterY = newcenterY/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
					newcenterZ = newcenterZ/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
					displacementX = newcenterX - generalParams.centerX;
					displacementY = newcenterY - generalParams.centerY;
					displacementZ = newcenterZ - generalParams.centerZ;
					
				//	std::cout<<"HERE3?"<<std::endl;
					for (int i = 0; i < generalParams.maxNodeCount; i++){
					coordInfoVecs.nodeLocX[i] += -displacementX;
					coordInfoVecs.nodeLocY[i] += -displacementY;
					coordInfoVecs.nodeLocZ[i] += -displacementZ;
					}
				//	std::cout<<"HERE4?"<<std::endl;
					for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){
						ljInfoVecs.LJ_PosX_all[i] += -displacementX;
						ljInfoVecs.LJ_PosY_all[i] += -displacementY;
						ljInfoVecs.LJ_PosZ_all[i] += -displacementZ;
					}

					ComputeVolume(
						generalParams,
						coordInfoVecs,
						linearSpringInfoVecs,
						ljInfoVecs);

				}

		}

		end_of_relaxation = true;

		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////Recalculate "u" since the system is relaxed, potentially becoming different ////////////////////
		////////////////////from the initial configuration /////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		max_cell_triangle_height = -10000.0;
		min_cell_triangle_height = 10000.0;
		for (int i = 0; i < coordInfoVecs.num_triangles; i++){
			if (coordInfoVecs.triangles2Nodes_1[i] >= (INT_MAX-100) || coordInfoVecs.triangles2Nodes_2[i] >= (INT_MAX-100) || coordInfoVecs.triangles2Nodes_3[i] >= (INT_MAX-100)){
				continue;
			}
			if (coordInfoVecs.triangles2Nodes_1[i] < 0 || coordInfoVecs.triangles2Nodes_2[i] < 0 || coordInfoVecs.triangles2Nodes_3[i] < 0){
				continue;
			}
			v1 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[i]];
			v2 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[i]];
			v3 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[i]];
			v4 = (v1+v2+v3)/3.0;
			if (v4 >= max_cell_triangle_height){
				max_cell_triangle_height = v4;
			}
			if (v4 <= min_cell_triangle_height){
				min_cell_triangle_height = v4;
			}
		}
		cell_height = (max_cell_triangle_height - min_cell_triangle_height);
		
		// double max_u = 3.0;//1.1;
		// double min_u = 0.05;//0.9;
		// vector<double> u;
		// coordInfoVecs.u.resize(3*coordInfoVecs.num_triangles);
		for (int i = 0; i < coordInfoVecs.num_triangles; i++){
			if (coordInfoVecs.triangles2Nodes_1[i] >= (INT_MAX-100) || coordInfoVecs.triangles2Nodes_2[i] >= (INT_MAX-100) || coordInfoVecs.triangles2Nodes_3[i] >= (INT_MAX-100)){
				
				continue;
			}
			if (coordInfoVecs.triangles2Nodes_1[i] < 0 || coordInfoVecs.triangles2Nodes_2[i] < 0 || coordInfoVecs.triangles2Nodes_3[i] < 0){
				continue;
			}
			v1 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[i]];
			v2 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[i]];
			v3 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[i]];
			v4 = (v1+v2+v3)/3.0;
			coordInfoVecs.u[i] = min_u + (max_u - min_u)*pow(((v4-min_cell_triangle_height)/cell_height),u_scalingPower); //Try exponential power? C_0*exp(-x/lambda), varying lambda
			// coordInfoVecs.u[i] = max_u - (max_u - min_u)*exp((-(v4-min_cell_triangle_height)/cell_height)/u_scalingPower);
		}
		
		if (polarization_postPeak == false){
			u_scalingPower = u_scalingPower+ u_scalingPower_Progress;
			if (u_scalingPower >= max_u_scalingPower){
				polarization_postPeak = true;
				u_scalingPower = max_u_scalingPower;
			}
		}
		else{
			u_scalingPower = u_scalingPower - u_scalingPower_postPeak_Progress;
			if (u_scalingPower <= max_u_scalingPower_postPeak){
				u_scalingPower = max_u_scalingPower_postPeak;
			}
		}
		
		// coordInfoVecs.beta = 1.0/(min_u + ((/*cell_height - */4.0*generalParams.Rmin)/cell_height)*(max_u - min_u));
	
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////Run chemical diffusion//////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		// generalParams.chemdiff_maxruntime = 2e3;
		int MAX_LOOP = 1;
		// if (edgeswap_iteration < 1){
			// utilities_ptr->LDG_Surface_Diffusion_Initialize(
			// 		coordInfoVecs,
			// 		generalParams,
			// 		build_ptr->hostSetInfoVecs,
			// 		auxVecs,
			// 		rbc,
			// 		n_rbc
			// );
		// }
		// if (needToRebuildDiffStructAfterEdgeSwap == true){
		// if (triggered == true){
			// utilities_ptr->LDG_Surface_Diffusion_Structure_Rebuild(coordInfoVecs,
			// 													generalParams,
			// 													build_ptr->hostSetInfoVecs,
			// 													auxVecs,
			// 													rbc,
			// 													n_rbc);
			// needToRebuildDiffStructAfterEdgeSwap = false;
		// }
		// if (triggered == true || edgeswap_iteration == 0){
			if (triggered == true){
				std::cout<<"Growth is triggered"<<std::endl;
			}
		if (edgeswap_iteration == 0 || edgeswap_iteration%GROWTH_FREQUENCY==0){
			std::cout<<"After rebuilding 'u', the new highest triangle is at the height of "<<max_cell_triangle_height<<std::endl;
			std::cout<<"new 1/beta calculated: "<< 1.0/coordInfoVecs.beta<<std::endl;
			utilities_ptr->LDG_Surface_Diffusion_Structure_Rebuild(coordInfoVecs,
																generalParams,
																build_ptr->hostSetInfoVecs,
																auxVecs,
																rbc,
																n_rbc);
			needToRebuildDiffStructAfterEdgeSwap = false;
			std::cout<<"Start solving PDE"<<std::endl;
			for (int i = 0; i < MAX_LOOP; i++){
				utilities_ptr->LDG_Surface_Diffusion_Solve(
									edgeswap_iteration,
									coordInfoVecs,
									generalParams,
									build_ptr->hostSetInfoVecs,
									auxVecs,
									rbc,
									n_rbc);
				// storage->storeVariables();
			}
			triggered = false;
		}
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
			
		if (end_of_relaxation == true){
			std::random_device rand_dev;
			// std::mt19937 generator2(rand_dev());
			std::mt19937 generator_edgeswap(rand_dev());
				ComputeVolume(
					generalParams,
					coordInfoVecs,
					linearSpringInfoVecs,
					ljInfoVecs);

				if ((generalParams.true_current_total_volume/initial_volume) < 0.6 || generalParams.true_current_total_volume/initial_volume >= MAX_VOLUME_RATIO){
					generalParams.true_num_edges = 0;
					for (int i = 0; i < coordInfoVecs.num_edges; i++){
						if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
							generalParams.true_num_edges += 1;
						}
					}
					storage-> print_VTK_File();
					storage-> storeVariables();
					
					if (generalParams.true_current_total_volume/initial_volume < 0.6){
						std::cout<<"Cell over compression 60%"<<std::endl;
					}
					else if (generalParams.true_current_total_volume/initial_volume >= MAX_VOLUME_RATIO){
						std::cout<<"Target volume ratio exceeded. Current volume ratio = "<<generalParams.true_current_total_volume/initial_volume<<std::endl;
					}
					std::cout<<"Current number of edgeswap iteration performed at volume-related termination = "<<edgeswap_iteration<<std::endl;
					std::cout<<"Current number of simulation step at volume-related termination = "<<number_of_simulation_step<<std::endl;

					Max_Runtime = 0.0;
					runSim = false;
					initial_kT = -1;
					break;
				}
					double current_bud_area = 0.0;
					for (int k = 0; k < coordInfoVecs.num_triangles; k++){
						if (coordInfoVecs.triangles2Nodes_1[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_1[k] <= (-INT_MAX + 1000.0) ||
							coordInfoVecs.triangles2Nodes_2[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_2[k] <= (-INT_MAX + 1000.0) ||
							coordInfoVecs.triangles2Nodes_3[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_3[k] <= (-INT_MAX + 1000.0)){
									continue;
								}
						else{
							if (generalParams.triangles_in_upperhem[k] == 1){
								double r1x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[k]];
								double r1y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[k]];
								double r1z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[k]];
								double r2x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_2[k]];
								double r2y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_2[k]];
								double r2z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[k]];
								double r3x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_3[k]];
								double r3y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_3[k]];
								double r3z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[k]];
								double norm_r1r2 = sqrt((r2x-r1x)*(r2x-r1x) + (r2y-r1y)*(r2y-r1y) + (r2z-r1z)*(r2z-r1z));
								double norm_r2r3 = sqrt((r3x-r2x)*(r3x-r2x) + (r3y-r2y)*(r3y-r2y) + (r3z-r2z)*(r3z-r2z));
								double norm_r3r1 = sqrt((r3x-r1x)*(r3x-r1x) + (r3y-r1y)*(r3y-r1y) + (r3z-r1z)*(r3z-r1z));
								double s = (norm_r1r2 + norm_r2r3 + norm_r3r1)/2.0;
								double area = sqrt(s*(s-norm_r1r2)*(s-norm_r2r3)*(s-norm_r3r1));
								current_bud_area += area;
							}
						}
					}
					// std::cout<<"Current bud surface area = "<<current_bud_area<<std::endl;
					if (current_bud_area/Initial_Bud_Area >= MAX_BUD_AREA_RATIO){
						std::cout<<"Target bud surface area ratio exceeded. Current bud surface area ratio = "<<current_bud_area/Initial_Bud_Area<<std::endl;
						std::cout<<"Current number of edgeswap iteration performed at area-related termination = "<<edgeswap_iteration<<std::endl;
						std::cout<<"Current number of simulation step at area-related termination = "<<number_of_simulation_step<<std::endl;
						generalParams.true_num_edges = 0;
						for (int i = 0; i < coordInfoVecs.num_edges; i++){
							if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
								generalParams.true_num_edges += 1;
							}
						}
						storage-> print_VTK_File();
						storage-> storeVariables();
						Max_Runtime = 0.0;
						runSim = false;
						initial_kT = -1;
						break;
					}

				// std::cout<<"entering edge swap algorithm"<<std::endl;							
				utilities_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
				
				// double max_conc = -INT_MAX;
				// double min_conc = INT_MAX;
				// double temp_max_conc, temp_min_conc;
				// for (int i = 0; i < coordInfoVecs.num_triangles; i++){
				// 	if (coordInfoVecs.triangles2Nodes_1[i] >= (INT_MAX - 1000) || coordInfoVecs.triangles2Nodes_1[i] <= (-INT_MAX + 1000)){
				// 		continue;
				// 	}
				// 	else if (coordInfoVecs.triangles2Nodes_2[i] >= (INT_MAX - 1000) || coordInfoVecs.triangles2Nodes_2[i] <= (-INT_MAX + 1000)){
				// 		continue;
				// 	}
				// 	else if (coordInfoVecs.triangles2Nodes_3[i] >= (INT_MAX - 1000) || coordInfoVecs.triangles2Nodes_3[i] <= (-INT_MAX + 1000)){
				// 		continue;
				// 	}

				// 	temp_max_conc = coordInfoVecs.soln_per_triangle[i];
				// 	if (temp_max_conc > max_conc){
				// 		max_conc = temp_max_conc;
				// 	}
				// 	if (temp_min_conc < min_conc){
				// 		min_conc = temp_min_conc;
				// 	}
				// }

				VectorShuffleForEdgeswapLoop.clear();
				for (int i = 0; i < coordInfoVecs.num_edges; i++){
					if (generalParams.edges_in_upperhem_list[i] >= 0 && 
						generalParams.edges_in_upperhem_list[i] != INT_MAX &&
						//generalParams.edges_in_upperhem[i] < coordInfoVecs.num_edges &&
						//generalParams.edges_in_upperhem[i] != -INT_MAX &&
						generalParams.boundaries_in_upperhem[i] != 1){
						// VectorShuffleForEdgeswapLoop.push_back(generalParams.edges_in_upperhem_list[i]);
						if (coordInfoVecs.edges2Nodes_1[i] < 0 || coordInfoVecs.edges2Nodes_1[i] >= (INT_MAX-1000)){
							continue;
						}
						else if (coordInfoVecs.edges2Nodes_2[i] < 0 || coordInfoVecs.edges2Nodes_2[i] >= (INT_MAX-1000)){
							continue;
						}
						// double tri1 = coordInfoVecs.edges2Triangles_1[i];
						// double tri2 = coordInfoVecs.edges2Triangles_2[i];
						// double avg_conc = (coordInfoVecs.soln_per_triangle[tri1] + coordInfoVecs.soln_per_triangle[tri2])/2.0;
						// if (avg_conc > max_conc_scaler_for_material_insert*max_conc){
						// 
							// VectorShuffleForGrowthLoop.push_back(y);
							// VectorShuffleForGrowthLoop_COUNT += 1;
							// VectorShuffleForEdgeswapLoop.push_back(i);
						// }
						VectorShuffleForEdgeswapLoop.push_back(i);
					}
				}	
				// std::cout<<"edge swap candidate vector size = "<<VectorShuffleForEdgeswapLoop.size()<<std::endl;
				num_edge_loop = round(true_num_edges_in_upperhem*SAMPLE_SIZE);
				if (num_edge_loop <= min_num_edge_loop){
					num_edge_loop = min_num_edge_loop;
				}
				
				std::shuffle(std::begin(VectorShuffleForEdgeswapLoop), std::end(VectorShuffleForEdgeswapLoop), generator_edgeswap);
				for (int edge_loop = 0; edge_loop < num_edge_loop; edge_loop++) {
													
					std::uniform_int_distribution<int> distribution(1,VectorShuffleForEdgeswapLoop.size());
					
					int dice_roll = distribution(generator_edgeswap);
					
					int edge = VectorShuffleForEdgeswapLoop[dice_roll - 1];
					//int edge = dice_roll -1;
					while (generalParams.boundaries_in_upperhem[edge] == 1 || edge == INT_MAX || edge < 0){
						dice_roll = distribution(generator_edgeswap);
						
						int edge = VectorShuffleForEdgeswapLoop[dice_roll - 1];
						//edge =  generalParams.edges_in_upperhem_list[dice_roll - 1];
						//edge = dice_roll -1;
						}
					//int edge = generalParams.edges_in_upperhem_list[edge_loop];
					//int edge = VectorShuffleForEdgeswapLoop[edge_loop];
					// std::cout<<"edge = "<<edge<<std::endl;
					if (edge < 0 || edge == INT_MAX){
						continue;
					}

					int ALPHA = utilities_ptr->edge_swap_host_vecs(
						edge,
						generalParams,
						build_ptr->hostSetInfoVecs,
						linearSpringInfoVecs,
						bendingTriangleInfoVecs,
						areaTriangleInfoVecs);
					if (ALPHA != 0){
						needToRebuildDiffStructAfterEdgeSwap = true;
					}
				}
			
			
			for (int i = 0; i < coordInfoVecs.num_triangles; i++){
				utilities_ptr->triangles2Triangles_host_vecs(i, build_ptr->hostSetInfoVecs,coordInfoVecs,generalParams, auxVecs);
			}
		
			utilities_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);//Currently this is treated as a backup of coordInfoVecs
	
			EDGESWAP_ALGORITHM_TRIGGERED = true;
			edgeswap_iteration += 1;
			translate_counter += 1;
		}
		
	
		if (EDGESWAP_ALGORITHM_TRIGGERED == false){
			//std::cout<<"current_time = "<<current_time<<std::endl;
			std::cout<<"EDGE_SWAP IS TRIGGERED BECAUSE PREVIOUS RELAXATION STEPS SOMEHOW FAIL TO TRIGGER EDGESWAP NORMALLY. PLEASE INVESTIGATE."<<std::endl;
			runSim = false;
			initial_kT = -1;
			Max_Runtime = 0.0;
			break;
		}
		
		if (edgeswap_iteration % (GROWTH_FREQUENCY*GROWTH_FREQUENCY_SCALE) == 0){
				for (int v = 0; v < coordInfoVecs.num_edges; v++){
				double ev1 = coordInfoVecs.edges2Nodes_1[v];
				double ev2 = coordInfoVecs.edges2Nodes_2[v];
				if (ev1 == INT_MAX || ev2 == INT_MAX){
					continue;
				}
				double ed = sqrt((coordInfoVecs.nodeLocX[ev2] - coordInfoVecs.nodeLocX[ev1])*(coordInfoVecs.nodeLocX[ev2] - coordInfoVecs.nodeLocX[ev1]) +
							(coordInfoVecs.nodeLocY[ev2] - coordInfoVecs.nodeLocY[ev1])*(coordInfoVecs.nodeLocY[ev2] - coordInfoVecs.nodeLocY[ev1]) +
							(coordInfoVecs.nodeLocZ[ev2] - coordInfoVecs.nodeLocZ[ev1])*(coordInfoVecs.nodeLocZ[ev2] - coordInfoVecs.nodeLocZ[ev1]));
				if (ed >= 2.0){
					std::cout<<"Edge over extension, possibly some instability occuring. Aborting the simulation."<<std::endl;
					runSim = false;
					initial_kT = -1;
					break;
				}
			}
			// generalParams.angle_per_edge.clear();
			generalParams.true_num_edges = 0;
			for (int i = 0; i < coordInfoVecs.num_edges; i++){
				if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
					generalParams.true_num_edges += 1;
				}
				}
				storage->print_VTK_File();
				storage->storeVariables();
				
				double current_bud_area = 0.0;
				for (int k = 0; k < coordInfoVecs.num_triangles; k++){
				if (coordInfoVecs.triangles2Nodes_1[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_1[k] <= (-INT_MAX + 1000.0) ||
					coordInfoVecs.triangles2Nodes_2[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_2[k] <= (-INT_MAX + 1000.0) ||
					coordInfoVecs.triangles2Nodes_3[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_3[k] <= (-INT_MAX + 1000.0)){
							continue;
						}
				else{
					if (generalParams.triangles_in_upperhem[k] == 1){
						double r1x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[k]];
						double r1y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[k]];
						double r1z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[k]];
						double r2x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_2[k]];
						double r2y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_2[k]];
						double r2z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[k]];
						double r3x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_3[k]];
						double r3y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_3[k]];
						double r3z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[k]];
						double norm_r1r2 = sqrt((r2x-r1x)*(r2x-r1x) + (r2y-r1y)*(r2y-r1y) + (r2z-r1z)*(r2z-r1z));
						double norm_r2r3 = sqrt((r3x-r2x)*(r3x-r2x) + (r3y-r2y)*(r3y-r2y) + (r3z-r2z)*(r3z-r2z));
						double norm_r3r1 = sqrt((r3x-r1x)*(r3x-r1x) + (r3y-r1y)*(r3y-r1y) + (r3z-r1z)*(r3z-r1z));
						double s = (norm_r1r2 + norm_r2r3 + norm_r3r1)/2.0;
						double area = sqrt(s*(s-norm_r1r2)*(s-norm_r2r3)*(s-norm_r3r1));
						current_bud_area += area;
					}
				}
				}
				std::cout<<"Current bud surface area = "<<current_bud_area<<std::endl;
				std::cout<<"Current number of edgeswap performed = "<<edgeswap_iteration<<std::endl;
			//  std::cout<<"current Hill equation constant = "<<generalParams.hilleqnconst<<std::endl;
				//storage->storeVariables();
				std::cout<<"current total energy = "<< new_total_energy<<std::endl;
			//  std::cout<<"LINEAR ENERGY = "<<linearSpringInfoVecs.linear_spring_energy<<std::endl;
			// std::cout<<"BEND ENERGY = "<<bendingTriangleInfoVecs.bending_triangle_energy<<std::endl;
			// std::cout<<"AREA ENERGY = "<<areaTriangleInfoVecs.area_triangle_energy<<std::endl;
			//std::cout<<"REPULSION ENERGY = "<<energy_rep<<std::endl;
			// std::cout<<"VOLUME ENERGY = "<<generalParams.volume_energy<<std::endl;
				std::cout<<"energy_gradient = "<<energy_gradient<<std::endl;
				std::cout<<"true current total volume = "<<generalParams.true_current_total_volume<<std::endl;
			std::cout<<"equilibrium total volume = "<<generalParams.eq_total_volume<<std::endl;
			std::cout<<"u_scalingPower updated to : "<<u_scalingPower<<std::endl;
		}
		if (edgeswap_iteration == NKBT-1 ){
			true_num_edges_in_upperhem = 0;
			for (int i = 0; i < coordInfoVecs.num_edges; i++){
				if (generalParams.edges_in_upperhem_list[i] != INT_MAX && generalParams.edges_in_upperhem_list[i] >= 0){
					true_num_edges_in_upperhem += 1;
					//break;
				}
			}
			storage->print_VTK_File();
			storage->storeVariables();
			std::cout<<"Allowed number of simulation steps reached. Simulation terminated."<<std::endl;
			runSim = false;
			initial_kT = -1;
			break;
			}

//std::cout<<"ERROR BEFORE GROWTH"<<std::endl;
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////// GROWTH OF THE CELL (MEMBRANE) ////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////	
		// GROWTH_COUNTER = 0;
		// if (1 < 0){
		// if (edgeswap_iteration % GROWTH_FREQUENCY == 0){
		if (edgeswap_iteration % GROWTH_FREQUENCY == 0 && TOTAL_GROWTH_ATTEMPT < NUMBER_OF_GROWTH_EVENT){
			std::cout<<"Entering growth algorithm"<<std::endl;
			GROWTH_COUNTER += 1;
			// VectorShuffleForGrowthLoop.clear();
			// int VectorShuffleForGrowthLoop_COUNT = 0;
			// max_height = -10000.0;
			// double current_center_x = 0.0;
			// double current_center_y = 0.0;

			// for (int k = 0; k < generalParams.maxNodeCount; k++){
			// 	if (generalParams.nodes_in_upperhem[k] == 1){
			// 		current_center_x += coordInfoVecs.nodeLocX[k];
			// 		current_center_y += coordInfoVecs.nodeLocX[k];
			// 	}
				
			// 	if (coordInfoVecs. nodeLocZ[k] >= max_height){
			// 		max_height = coordInfoVecs.nodeLocZ[k];
			// 		max_height_index = k;
			// 	}

			// }
			// current_center_x = current_center_x/generalParams.maxNodeCount;
			// current_center_y = current_center_y/generalParams.maxNodeCount;
			// // double bdry_to_tip = 0.0;
			// double bdry_to_tip_height = 0.0;
			// for (int y = 0; y < boundary_edge_list.size(); y++){
			// 	// double edge_mdpt_x = (coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_1[boundary_edge_list[y]]] +
			// 	// 						coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_2[boundary_edge_list[y]]])/2.0;
			// 	// double edge_mdpt_y = (coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_1[boundary_edge_list[y]]] +
			// 	// 						coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_2[boundary_edge_list[y]]])/2.0;
			// 	double edge_mdpt_z = (coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_1[boundary_edge_list[y]]] +
			// 							coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_2[boundary_edge_list[y]]])/2.0;
			// 	// bdry_to_tip += sqrt(pow(current_center_x - edge_mdpt_x,2.0)+pow(current_center_y - edge_mdpt_y,2.0)+pow(coordInfoVecs.nodeLocZ[max_height_index] - edge_mdpt_z,2.0));
			// 	bdry_to_tip_height += sqrt(pow(coordInfoVecs.nodeLocZ[max_height_index] - edge_mdpt_z,2.0));
			// }
			// // bdry_to_tip = bdry_to_tip/boundary_edge_list.size();
			// bdry_to_tip_height = bdry_to_tip_height/boundary_edge_list.size();
			double max_conc = -INT_MAX;
			double min_conc = INT_MAX;
			double temp_max_conc, temp_min_conc;
			int max_conc_idx;
			for (int i = 0; i < coordInfoVecs.num_triangles; i++){
				if (coordInfoVecs.triangles2Nodes_1[i] >= (INT_MAX - 1000) || coordInfoVecs.triangles2Nodes_1[i] <= (-INT_MAX + 1000)){
					continue;
				}
				else if (coordInfoVecs.triangles2Nodes_2[i] >= (INT_MAX - 1000) || coordInfoVecs.triangles2Nodes_2[i] <= (-INT_MAX + 1000)){
					continue;
				}
				else if (coordInfoVecs.triangles2Nodes_3[i] >= (INT_MAX - 1000) || coordInfoVecs.triangles2Nodes_3[i] <= (-INT_MAX + 1000)){
					continue;
				}

				temp_max_conc = coordInfoVecs.soln_per_triangle[i];
				if (temp_max_conc > max_conc){
					max_conc = temp_max_conc;
					max_conc_idx = i;
				}
				if (temp_min_conc < min_conc){
					min_conc = temp_min_conc;
				}
			}
			std::cout<<"max_conc = "<<max_conc<<std::endl;
			std::cout<<"min_conc = "<<min_conc<<std::endl;
			int max_conc_triNode1 = coordInfoVecs.triangles2Nodes_1[max_conc_idx];
			int max_conc_triNode2 = coordInfoVecs.triangles2Nodes_2[max_conc_idx];
			int max_conc_triNode3 = coordInfoVecs.triangles2Nodes_3[max_conc_idx];
			std::cout<<"location of max conc triangle: "<<coordInfoVecs.nodeLocX[max_conc_triNode1]<<", "<<coordInfoVecs.nodeLocY[max_conc_triNode1]<<", "<<coordInfoVecs.nodeLocZ[max_conc_triNode1]<<std::endl;
			std::cout<<"location of max conc triangle: "<<coordInfoVecs.nodeLocX[max_conc_triNode2]<<", "<<coordInfoVecs.nodeLocY[max_conc_triNode2]<<", "<<coordInfoVecs.nodeLocZ[max_conc_triNode2]<<std::endl;
			std::cout<<"location of max conc triangle: "<<coordInfoVecs.nodeLocX[max_conc_triNode3]<<", "<<coordInfoVecs.nodeLocY[max_conc_triNode3]<<", "<<coordInfoVecs.nodeLocZ[max_conc_triNode3]<<std::endl;

			VectorShuffleForGrowthLoop.clear();
			int VectorShuffleForGrowthLoop_COUNT = 0;

			for (int y = 0; y < coordInfoVecs.num_edges; y++){
				// std::cout<<y<<std::endl;
				if (generalParams.edges_in_upperhem_list[y] >= 0 &&
					generalParams.edges_in_upperhem_list[y] != INT_MAX &&
					generalParams.edges_in_upperhem_list[y] <= (INT_MAX-1000) &&
					generalParams.edges_in_upperhem_list[y] >= (-INT_MAX+1000) &&
					generalParams.boundaries_in_upperhem[y] != 1){
						// std::cout<<"IF condition satisfied"<<std::endl;
						// std::cout<<"generalParams.edges_in_upperhem_list = "<<generalParams.edges_in_upperhem_list[y]<<std::endl;
						if (coordInfoVecs.edges2Nodes_1[y] < 0 || coordInfoVecs.edges2Nodes_1[y] >= (INT_MAX-1000)){
							continue;
						}
						else if (coordInfoVecs.edges2Nodes_2[y] < 0 || coordInfoVecs.edges2Nodes_2[y] >= (INT_MAX-1000)){
							continue;
						}
						// double edge_mdpt_x = (coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_1[y]] +
						// 					coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_2[y]])/2.0;
						// // std::cout<<edge_mdpt_x<<std::endl;
						// double edge_mdpt_y = (coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_1[y]] +
						// 						coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_2[y]])/2.0;
						// // std::cout<<edge_mdpt_y<<std::endl;
						// double edge_mdpt_z = (coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_1[y]] +
						// 						coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_2[y]])/2.0;
						// // std::cout<<edge_mdpt_z<<std::endl;
						// // double current_edge_to_tip = sqrt(pow(current_center_x - edge_mdpt_x,2.0)+pow(current_center_y - edge_mdpt_y,2.0)+pow(coordInfoVecs.nodeLocZ[max_height_index] - edge_mdpt_z,2.0));
						// double current_edge_to_tip_height = sqrt(pow(coordInfoVecs.nodeLocZ[max_height_index] - edge_mdpt_z,2.0));
						// double current_edge_to_tip_dist = sqrt(pow(coordInfoVecs.nodeLocX[max_height_index] - edge_mdpt_x,2.0) + pow(coordInfoVecs.nodeLocY[max_height_index] - edge_mdpt_y,2.0) + pow(coordInfoVecs.nodeLocZ[max_height_index] - edge_mdpt_z,2.0));
						// std::cout<<"current_edge_to_tip = "<<current_edge_to_tip<<std::endl;
					// if ((current_edge_to_tip/bdry_to_tip) <= 0.8 && bdry_to_tip >= (dtb*1.5)){
					// if ((current_edge_to_tip_height/bdry_to_tip_height) <= 0.33 && bdry_to_tip_height >= (dtb*1.5)){
					// if ((current_edge_to_tip_height) <= dtb*1.0 && bdry_to_tip_height >= (dtb*1.5)){
					// if ((current_edge_to_tip_height) <= generalParams.Rmin*current_edge_to_tip_height_scale && bdry_to_tip_height >= (generalParams.Rmin*bdry_to_tip_height_scale)){
					double tri1 = coordInfoVecs.edges2Triangles_1[y];
					double tri2 = coordInfoVecs.edges2Triangles_2[y];
					double avg_conc = (coordInfoVecs.soln_per_triangle[tri1] + coordInfoVecs.soln_per_triangle[tri2])/2.0;
					if (avg_conc > max_conc_scaler_for_material_insert*max_conc){
					// if (avg_conc > (max_conc - 0.1*(max_conc - min_conc))){
						VectorShuffleForGrowthLoop.push_back(y);
						VectorShuffleForGrowthLoop_COUNT += 1;
						// if (VectorShuffleForEdgeswapLoop.size() == 1){
						// 	std::cout<<"avg_conc = "<<avg_conc<<std::endl;
						// }
					}
					// if ((current_edge_to_tip_dist) <= generalParams.Rmin*current_edge_to_tip_dist_scale && bdry_to_tip_height >= (generalParams.Rmin*bdry_to_tip_height_scale)){
					// 	VectorShuffleForGrowthLoop.push_back(y);
					// 	VectorShuffleForGrowthLoop_COUNT += 1;
					// }
					// else if(bdry_to_tip < (dtb*1.5)){
					// else if(bdry_to_tip_height < (generalParams.Rmin*bdry_to_tip_height_scale)){
					// 	VectorShuffleForGrowthLoop.push_back(y);
					// 	VectorShuffleForGrowthLoop_COUNT += 1;
					// }
				}
				/*if (generalParams.edges_in_upperhem_list[y] >= 0 &&
					generalParams.edges_in_upperhem_list[y] != INT_MAX &&
					generalParams.boundaries_in_upperhem[y] != 1 &&
					edges_in_growth[y] == 1){
					VectorShuffleForGrowthLoop.push_back(y);
				}*/
				
				
			}
			
			// for (int y = 0; y < coordInfoVecs.num_edges; y++){
			// 	if (generalParams.edges_in_upperhem_list[y] >= 0 &&
			// 		generalParams.edges_in_upperhem_list[y] != INT_MAX &&
			// 		generalParams.boundaries_in_upperhem[y] != 1){
			// 		VectorShuffleForGrowthLoop.push_back(y);
			// 		VectorShuffleForGrowthLoop_COUNT += 1;
			// 	}
			// 	/*if (generalParams.edges_in_upperhem_list[y] >= 0 &&
			// 		generalParams.edges_in_upperhem_list[y] != INT_MAX &&
			// 		generalParams.boundaries_in_upperhem[y] != 1 &&
			// 		edges_in_growth[y] == 1){
			// 		VectorShuffleForGrowthLoop.push_back(y);
			// 	}*/
				
				
			// }
			std::cout<<"VectorShuffleForGrowthLoop_COUNT = "<<VectorShuffleForGrowthLoop_COUNT<<std::endl;

			std::random_device rand_dev;
			std::mt19937 generator3(rand_dev());
			std::shuffle(std::begin(VectorShuffleForGrowthLoop), std::end(VectorShuffleForGrowthLoop), generator3);
			int MAX_GROWTH_TEST = VectorShuffleForGrowthLoop.size();
			// bool triggered = false;
			int true_DELTA = 0;
			int MAX_GROWTH_PER_GROWTH_EVENT = 1;

			generalParams.triangle_undergoing_growth.clear();

			std::cout<<"BEGIN GROWTH ALGORITHM"<<std::endl;
			utilities_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
			int GROWTH_COUNT = 0;
			for (int p = 0; p < MAX_GROWTH_TEST; p++){
				if (coordInfoVecs.edges2Nodes_1[VectorShuffleForGrowthLoop[p]] < 0 || coordInfoVecs.edges2Nodes_1[VectorShuffleForGrowthLoop[p]] == INT_MAX){
					continue;
				}
				else if (coordInfoVecs.edges2Nodes_2[VectorShuffleForGrowthLoop[p]] < 0 || coordInfoVecs.edges2Nodes_2[VectorShuffleForGrowthLoop[p]] == INT_MAX){
					continue;
				}
				//std::cout<<"begin growth test"<<std::endl;
				int DELTA = utilities_ptr->growth_host_vecs(
					VectorShuffleForGrowthLoop[p],
					generalParams,
					build_ptr->hostSetInfoVecs,
					coordInfoVecs,
					linearSpringInfoVecs,
					bendingTriangleInfoVecs,
					areaTriangleInfoVecs);
				if (DELTA >= 0){
					double tri1 = coordInfoVecs.edges2Triangles_1[DELTA];
					double tri2 = coordInfoVecs.edges2Triangles_2[DELTA];
					double avg_conc = (coordInfoVecs.soln_per_triangle[tri1] + coordInfoVecs.soln_per_triangle[tri2])/2.0;
					std::cout<<" edge chosen for growth has avg_conc of : "<<avg_conc<<std::endl;
					GROWTH_COUNT += 1;
					TOTAL_GROWTH_COUNTER += 1;
				}
				else{
					generalParams.triangle_undergoing_growth.clear();
				}

				if (GROWTH_COUNT >= MAX_GROWTH_PER_GROWTH_EVENT){
					break;
				}
			}
			TOTAL_GROWTH_ATTEMPT += 1;
			utilities_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
			std::cout<<"END GROWTH ALGORITHM"<<std::endl;
			
			if (generalParams.triangle_undergoing_growth.size() > 1){
				triggered = true;
				std::cout<<"Begin rebuilding data structure post growth"<<std::endl;
					// // This means that growth actually happened. Time to rebuild the data structure and update
					// //the chemical concentration data structure as well.
					utilities_ptr->LDG_Surface_Diffusion_Structure_Rebuild_postGrowth(coordInfoVecs,
																			generalParams,
																			build_ptr->hostSetInfoVecs,
																			auxVecs,
																			rbc,
																			n_rbc);
				std::cout<<"End rebuilding data structure post growth"<<std::endl;
			//	storage->print_VTK_File();
			//	storage->storeVariables();
			}
			
			std::cout<<"number of cell wall insertion = "<<GROWTH_COUNT<<std::endl;
			std::cout<<"Total growth event triggered = "<<TOTAL_GROWTH_COUNTER<<std::endl;
			std::cout<<"Total growth event attempt = "<<TOTAL_GROWTH_ATTEMPT<<std::endl;
				// if (triggered == true){	
				// 	true_num_edges_in_upperhem = 0;
				// 	for (int i = 0; i < coordInfoVecs.num_edges; i++){
				// 		if (generalParams.edges_in_upperhem_list[i] != INT_MAX && generalParams.edges_in_upperhem_list[i] >= 0){
				// 			true_num_edges_in_upperhem += 1;
				// 			//break;
				// 		}
				// 	}
				// 	//std::cout<<"WHERE iS THE PROBLEM 3"<<std::endl;
				// }
				std::cout<<"Exiting growth algorithm"<<std::endl;
			}	
		}
	}
};
	
	
	





void System::assignStorage(std::shared_ptr<Storage> _storage) {
	storage = _storage;
};
void System::set_weak_builder(std::weak_ptr<SystemBuilder> _weak_bld_ptr) {
	weak_bld_ptr = _weak_bld_ptr;
};



//initialize memory for thrust vectors and set coordInfoVecs vals from input. 
void System::initializeSystem(HostSetInfoVecs& hostSetInfoVecs) {
	std::cout<<"Initializing"<<std::endl;

	generalParams.maxNodeCount = hostSetInfoVecs.nodeLocX.size();
	coordInfoVecs.num_edges = hostSetInfoVecs.edges2Nodes_1.size();
	coordInfoVecs.num_triangles = hostSetInfoVecs.triangles2Nodes_1.size();

	std::cout<<"num nodes: "<< generalParams.maxNodeCount << std::endl;
	std::cout<<"num edges: "<< coordInfoVecs.num_edges << std::endl;
	std::cout<<"num elems: "<< coordInfoVecs.num_triangles << std::endl;
	//allocate memory
	int mem_prealloc = 4;
	coordInfoVecs.scaling_per_edge.resize(mem_prealloc*coordInfoVecs.num_edges, 0.0);
	hostSetInfoVecs.scaling_per_edge.resize(coordInfoVecs.scaling_per_edge.size(), 0.0);

	coordInfoVecs.soln_per_triangle.resize(mem_prealloc*coordInfoVecs.num_triangles, INT_MAX);
	coordInfoVecs.b_per_triangle.resize(mem_prealloc*coordInfoVecs.num_triangles, INT_MAX);

	coordInfoVecs.isNodeFixed.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(),false);
	coordInfoVecs.prevNodeLocX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeLocY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeLocZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());

	coordInfoVecs.prevNodeForceX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeForceY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeForceZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	
	coordInfoVecs.nodeLocX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.nodeLocY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.nodeLocZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());

	coordInfoVecs.nodeForceX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(), 0.0);
	coordInfoVecs.nodeForceY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(), 0.0);
	coordInfoVecs.nodeForceZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(), 0.0);

	coordInfoVecs.triangles2Nodes_1.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Nodes_2.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Nodes_3.resize( mem_prealloc*coordInfoVecs.num_triangles );
	
	coordInfoVecs.triangles2Edges_1.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Edges_2.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Edges_3.resize( mem_prealloc*coordInfoVecs.num_triangles );

	coordInfoVecs.triangles2Triangles_1.resize( mem_prealloc*coordInfoVecs.num_triangles, -INT_MAX );
	coordInfoVecs.triangles2Triangles_2.resize( mem_prealloc*coordInfoVecs.num_triangles, -INT_MAX );
	coordInfoVecs.triangles2Triangles_3.resize( mem_prealloc*coordInfoVecs.num_triangles, -INT_MAX );

	hostSetInfoVecs.triangles2Triangles_1.resize( mem_prealloc*coordInfoVecs.num_triangles, -INT_MAX );
	hostSetInfoVecs.triangles2Triangles_2.resize( mem_prealloc*coordInfoVecs.num_triangles, -INT_MAX );
	hostSetInfoVecs.triangles2Triangles_3.resize( mem_prealloc*coordInfoVecs.num_triangles, -INT_MAX );

	coordInfoVecs.edges2Nodes_1.resize( mem_prealloc*coordInfoVecs.num_edges );
	coordInfoVecs.edges2Nodes_2.resize( mem_prealloc*coordInfoVecs.num_edges );
	
	coordInfoVecs.edges2Triangles_1.resize( mem_prealloc*coordInfoVecs.num_edges );
	coordInfoVecs.edges2Triangles_2.resize( mem_prealloc*coordInfoVecs.num_edges );

	coordInfoVecs.nndata1.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata2.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata3.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata4.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata5.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata6.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata7.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata8.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata9.resize( mem_prealloc*generalParams.maxNodeCount);
	//coordInfoVecs.nndata10.resize( mem_prealloc*generalParams.maxNodeCount);
	//coordInfoVecs.nndata11.resize( mem_prealloc*generalParams.maxNodeCount);
	//coordInfoVecs.nndata12.resize( mem_prealloc*generalParams.maxNodeCount);

	coordInfoVecs.SurfaceNormalX.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.SurfaceNormalY.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.SurfaceNormalZ.resize( mem_prealloc*generalParams.maxNodeCount);

	generalParams.nodes_in_upperhem.resize(mem_prealloc*generalParams.maxNodeCount);
	generalParams.triangles_in_upperhem.resize(mem_prealloc*coordInfoVecs.num_triangles);
	generalParams.edges_in_upperhem.resize(mem_prealloc*coordInfoVecs.num_edges);
	generalParams.edges_in_upperhem_list.resize(mem_prealloc*coordInfoVecs.num_edges);
	generalParams.boundaries_in_upperhem.resize(mem_prealloc*coordInfoVecs.num_edges, -1);

	hostSetInfoVecs.nodes_in_upperhem.resize(generalParams.nodes_in_upperhem.size());
	hostSetInfoVecs.triangles_in_upperhem.resize(generalParams.triangles_in_upperhem.size());
	hostSetInfoVecs.edges_in_upperhem.resize(generalParams.edges_in_upperhem.size());
	hostSetInfoVecs.edges_in_upperhem_list.resize(mem_prealloc*coordInfoVecs.num_edges);
	hostSetInfoVecs.boundaries_in_upperhem.resize(mem_prealloc*coordInfoVecs.num_edges, -1);

	hostSetInfoVecs.nodes2Triangles_1.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_2.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_3.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_4.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_5.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_6.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_7.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_8.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_9.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	

	coordInfoVecs.nodes2Triangles_1.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_2.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_3.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_4.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_5.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_6.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_7.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_8.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_9.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	

	thrust::copy(coordInfoVecs.nodes2Triangles_1.begin(), coordInfoVecs.nodes2Triangles_1.end(), hostSetInfoVecs.nodes2Triangles_1.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_2.begin(), coordInfoVecs.nodes2Triangles_2.end(), hostSetInfoVecs.nodes2Triangles_2.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_3.begin(), coordInfoVecs.nodes2Triangles_3.end(), hostSetInfoVecs.nodes2Triangles_3.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_4.begin(), coordInfoVecs.nodes2Triangles_4.end(), hostSetInfoVecs.nodes2Triangles_4.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_5.begin(), coordInfoVecs.nodes2Triangles_5.end(), hostSetInfoVecs.nodes2Triangles_5.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_6.begin(), coordInfoVecs.nodes2Triangles_6.end(), hostSetInfoVecs.nodes2Triangles_6.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_7.begin(), coordInfoVecs.nodes2Triangles_7.end(), hostSetInfoVecs.nodes2Triangles_7.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_8.begin(), coordInfoVecs.nodes2Triangles_8.end(), hostSetInfoVecs.nodes2Triangles_8.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_9.begin(), coordInfoVecs.nodes2Triangles_9.end(), hostSetInfoVecs.nodes2Triangles_9.begin() );
	//thrust::copy(coordInfoVecs.nodes2Triangles_10.begin(), coordInfoVecs.nodes2Triangles_10.end(), hostInfoVecs.nodes2Triangles_10.begin() );
	//thrust::copy(coordInfoVecs.nodes2Triangles_11.begin(), coordInfoVecs.nodes2Triangles_11.end(), hostInfoVecs.nodes2Triangles_11.begin() );
	//thrust::copy(coordInfoVecs.nodes2Triangles_12.begin(), coordInfoVecs.nodes2Triangles_12.end(), hostInfoVecs.nodes2Triangles_12.begin() );

	//copy info to GPU
	std::cout<<"Copying"<<std::endl;
	thrust::copy(hostSetInfoVecs.isNodeFixed.begin(),hostSetInfoVecs.isNodeFixed.end(), coordInfoVecs.isNodeFixed.begin());
	
	std::cout<<"fixed_node_in_host: "<<std::endl;
	for (int k = 0; k < hostSetInfoVecs.isNodeFixed.size(); k++){
		//std::cout<<hostSetInfoVecs.isNodeFixed[k]<<std::endl;
	}
	std::cout<<"end_of_fixed_node_host_printout"<<std::endl;
	std::cout<<"fixed_node_in_device: "<<std::endl;
	for (int k = 0; k < coordInfoVecs.isNodeFixed.size(); k++){
		//std::cout<<coordInfoVecs.isNodeFixed[k]<<std::endl;
	}
	std::cout<<"end_of_fixed_node_device_printout"<<std::endl;
std::cout<<"size of host fixed "<< hostSetInfoVecs.isNodeFixed.size()<<std::endl;
std::cout<<"size of device fixed "<< coordInfoVecs.isNodeFixed.size()<<std::endl;

	/*for (int k = 0; k < coordInfoVecs.isNodeFixed.size(); k++){
		bool isFixedHost = hostSetInfoVecs.isNodeFixed[k];
		bool isFixedDevice = coordInfoVecs.isNodeFixed[k];
		if (isFixedDevice != isFixedHost){

			std::cout<<"pos "<< k << " dev val = " << coordInfoVecs.isNodeFixed[k]
				<< " host val = " <<  hostSetInfoVecs.isNodeFixed[k] <<std::endl;
		}
	}*/
	thrust::fill(coordInfoVecs.nodeForceX.begin(), coordInfoVecs.nodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceY.begin(), coordInfoVecs.nodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceZ.begin(), coordInfoVecs.nodeForceZ.end(), 0.0);

	thrust::fill(coordInfoVecs.prevNodeForceX.begin(), coordInfoVecs.prevNodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.prevNodeForceY.begin(), coordInfoVecs.prevNodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.prevNodeForceZ.begin(), coordInfoVecs.prevNodeForceZ.end(), 0.0);
	
	thrust::copy(hostSetInfoVecs.nodeLocX.begin(), hostSetInfoVecs.nodeLocX.end(), coordInfoVecs.prevNodeLocX.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocY.begin(), hostSetInfoVecs.nodeLocY.end(), coordInfoVecs.prevNodeLocY.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocZ.begin(), hostSetInfoVecs.nodeLocZ.end(), coordInfoVecs.prevNodeLocZ.begin() );
	
	thrust::copy(hostSetInfoVecs.nodeLocX.begin(), hostSetInfoVecs.nodeLocX.end(), coordInfoVecs.nodeLocX.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocY.begin(), hostSetInfoVecs.nodeLocY.end(), coordInfoVecs.nodeLocY.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocZ.begin(), hostSetInfoVecs.nodeLocZ.end(), coordInfoVecs.nodeLocZ.begin() );
	
	thrust::copy(hostSetInfoVecs.triangles2Nodes_1.begin(), hostSetInfoVecs.triangles2Nodes_1.end(), coordInfoVecs.triangles2Nodes_1.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Nodes_2.begin(), hostSetInfoVecs.triangles2Nodes_2.end(), coordInfoVecs.triangles2Nodes_2.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Nodes_3.begin(), hostSetInfoVecs.triangles2Nodes_3.end(), coordInfoVecs.triangles2Nodes_3.begin() );
	
	thrust::copy(hostSetInfoVecs.triangles2Edges_1.begin(), hostSetInfoVecs.triangles2Edges_1.end(), coordInfoVecs.triangles2Edges_1.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Edges_2.begin(), hostSetInfoVecs.triangles2Edges_2.end(), coordInfoVecs.triangles2Edges_2.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Edges_3.begin(), hostSetInfoVecs.triangles2Edges_3.end(), coordInfoVecs.triangles2Edges_3.begin() );

	thrust::copy(hostSetInfoVecs.edges2Nodes_1.begin(), hostSetInfoVecs.edges2Nodes_1.end(), coordInfoVecs.edges2Nodes_1.begin() );
	thrust::copy(hostSetInfoVecs.edges2Nodes_2.begin(), hostSetInfoVecs.edges2Nodes_2.end(), coordInfoVecs.edges2Nodes_2.begin() );
	
	thrust::copy(hostSetInfoVecs.edges2Triangles_1.begin(), hostSetInfoVecs.edges2Triangles_1.end(), coordInfoVecs.edges2Triangles_1.begin() );
	thrust::copy(hostSetInfoVecs.edges2Triangles_2.begin(), hostSetInfoVecs.edges2Triangles_2.end(), coordInfoVecs.edges2Triangles_2.begin() );

	thrust::copy(hostSetInfoVecs.nndata1.begin(), hostSetInfoVecs.nndata1.end(), coordInfoVecs.nndata1.begin() );
	thrust::copy(hostSetInfoVecs.nndata2.begin(), hostSetInfoVecs.nndata2.end(), coordInfoVecs.nndata2.begin() );
	thrust::copy(hostSetInfoVecs.nndata3.begin(), hostSetInfoVecs.nndata3.end(), coordInfoVecs.nndata3.begin() );
	thrust::copy(hostSetInfoVecs.nndata4.begin(), hostSetInfoVecs.nndata4.end(), coordInfoVecs.nndata4.begin() );
	thrust::copy(hostSetInfoVecs.nndata5.begin(), hostSetInfoVecs.nndata5.end(), coordInfoVecs.nndata5.begin() );
	thrust::copy(hostSetInfoVecs.nndata6.begin(), hostSetInfoVecs.nndata6.end(), coordInfoVecs.nndata6.begin() );
	thrust::copy(hostSetInfoVecs.nndata7.begin(), hostSetInfoVecs.nndata7.end(), coordInfoVecs.nndata7.begin() );
	thrust::copy(hostSetInfoVecs.nndata8.begin(), hostSetInfoVecs.nndata8.end(), coordInfoVecs.nndata8.begin() );
	thrust::copy(hostSetInfoVecs.nndata9.begin(), hostSetInfoVecs.nndata9.end(), coordInfoVecs.nndata9.begin() );
	//thrust::copy(hostSetInfoVecs.nndata10.begin(), hostSetInfoVecs.nndata10.end(), coordInfoVecs.nndata10.begin() );
	//thrust::copy(hostSetInfoVecs.nndata11.begin(), hostSetInfoVecs.nndata11.end(), coordInfoVecs.nndata11.begin() );
	//thrust::copy(hostSetInfoVecs.nndata12.begin(), hostSetInfoVecs.nndata12.end(), coordInfoVecs.nndata12.begin() );

	coordInfoVecs.u.resize(mem_prealloc*coordInfoVecs.num_triangles);


 
	//allocate memory for other data structures.   

	//area triangle info vec
	//number of area springs is the number of triangles
	std::cout<<"Mem"<<std::endl;
	areaTriangleInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	// std::cout<<"HERE 1?"<<std::endl;
	areaTriangleInfoVecs.tempNodeIdReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceXReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceYReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceZReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	// std::cout<<"HERE 2?"<<std::endl;
	//beinding triangle info vec
	//num bending springs is the number of times each edge is between two triangles. 
	bendingTriangleInfoVecs.numBendingSprings = coordInfoVecs.num_edges;//coordInfoVecs.edges2Triangles_1.size();
	// std::cout<<"HERE 2.5?"<<std::endl;
	bendingTriangleInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	// std::cout<<"HERE 3?"<<std::endl;
	bendingTriangleInfoVecs.tempNodeIdReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceXReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceYReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceZReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	// std::cout<<"HERE 4?"<<std::endl;
	//linear springs
	
	linearSpringInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	// std::cout<<"HERE 5?"<<std::endl;
	linearSpringInfoVecs.tempNodeIdReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceXReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceYReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceZReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	// std::cout<<"HERE 6?"<<std::endl;
	linearSpringInfoVecs.edge_initial_length.clear();
	//linearSpringInfoVecs.edge_initial_length.resize(mem_prealloc*coordInfoVecs.num_edges,1.0);
	
	//thrust::copy(hostSetInfoVecs.edge_initial_length.begin(), hostSetInfoVecs.edge_initial_length.end(), linearSpringInfoVecs.edge_initial_length.begin() );

	//Resize the hostSetInfoVecs so that we can copy data back and forth between hostSetinfoVecs and coordInfoVecs without problem.
	hostSetInfoVecs.isNodeFixed.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeLocX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeLocY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeLocZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());

	//hostSetInfoVecs.prevNodeForceX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeForceY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeForceZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	
	hostSetInfoVecs.nodeLocX.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeLocY.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeLocZ.resize(coordInfoVecs.nodeLocX.size());
	std::cout<<"Host_nodeLocX size = "<<hostSetInfoVecs.nodeLocX.size()<<std::endl;

	hostSetInfoVecs.nodeForceX.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeForceY.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeForceZ.resize(coordInfoVecs.nodeLocX.size());
	std::cout<<"Host_nodeForceX size = "<<hostSetInfoVecs.nodeLocX.size()<<std::endl;

	hostSetInfoVecs.triangles2Nodes_1.resize( coordInfoVecs.triangles2Nodes_1.size() );
	hostSetInfoVecs.triangles2Nodes_2.resize( coordInfoVecs.triangles2Nodes_2.size() );
	hostSetInfoVecs.triangles2Nodes_3.resize( coordInfoVecs.triangles2Nodes_3.size() );
	std::cout<<"Host_triangles2Nodes size = "<<hostSetInfoVecs.triangles2Nodes_1.size()<<std::endl;
	
	hostSetInfoVecs.triangles2Edges_1.resize( coordInfoVecs.triangles2Edges_1.size() );
	hostSetInfoVecs.triangles2Edges_2.resize( coordInfoVecs.triangles2Edges_2.size() );
	hostSetInfoVecs.triangles2Edges_3.resize( coordInfoVecs.triangles2Edges_3.size() );
	std::cout<<"Host_triangles2Edges size = "<<hostSetInfoVecs.triangles2Edges_1.size()<<std::endl;

	hostSetInfoVecs.edges2Nodes_1.resize( coordInfoVecs.edges2Nodes_1.size() );
	hostSetInfoVecs.edges2Nodes_2.resize( coordInfoVecs.edges2Nodes_2.size() );
	std::cout<<"Host_edges2Nodes size = "<<hostSetInfoVecs.edges2Nodes_1.size()<<std::endl;
	
	hostSetInfoVecs.edges2Triangles_1.resize( coordInfoVecs.edges2Triangles_1.size() );
	hostSetInfoVecs.edges2Triangles_2.resize( coordInfoVecs.edges2Triangles_2.size() );
	std::cout<<"Host_edges2Triangles size = "<<hostSetInfoVecs.edges2Triangles_1.size()<<std::endl;

	hostSetInfoVecs.nndata1.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata2.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata3.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata4.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata5.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata6.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata7.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata8.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata9.resize( mem_prealloc*generalParams.maxNodeCount);
	//hostSetInfoVecs.nndata10.resize( mem_prealloc*generalParams.maxNodeCount);
	//hostSetInfoVecs.nndata11.resize( mem_prealloc*generalParams.maxNodeCount);
	//hostSetInfoVecs.nndata12.resize( mem_prealloc*generalParams.maxNodeCount);

	//std::cout<<"initial lengths: "<< linearSpringInfoVecs.edge_initial_length.size()<<std::endl;

	std::cout<<"System Ready"<<std::endl;

	//Generate LJ particle list. and set LJ particle midpoint.
	//double maxX_lj = *(thrust::max_element(coordInfoVecs.nodeLocX.begin(),coordInfoVecs.nodeLocX.end()));
	//double minX_lj = *(thrust::min_element(coordInfoVecs.nodeLocX.begin(),coordInfoVecs.nodeLocX.end()));
	//double maxY_lj = *(thrust::max_element(coordInfoVecs.nodeLocY.begin(),coordInfoVecs.nodeLocY.end()));
	//double minY_lj = *(thrust::min_element(coordInfoVecs.nodeLocY.begin(),coordInfoVecs.nodeLocY.end()));
	
	//ljInfoVecs.LJ_PosX = (maxX_lj + minX_lj)/2.0;
	//ljInfoVecs.LJ_PosY = (maxY_lj + minY_lj)/2.0;


	//currently unused
	/*thrust::host_vector<int> tempIds;
	for (int i = 0; i < hostSetInfoVecs.nodeLocX.size(); i++ ) {
		double xLoc = hostSetInfoVecs.nodeLocX[i];
		double yLoc = hostSetInfoVecs.nodeLocY[i];
		double zLoc = hostSetInfoVecs.nodeLocZ[i];
		
		double xDist = ljInfoVecs.LJ_PosX - xLoc;
		double yDist = ljInfoVecs.LJ_PosY - yLoc;
		double zDist = ljInfoVecs.LJ_PosZ - zLoc;

		double dist = std::sqrt(xDist*xDist + yDist*yDist + zDist*zDist);
		//just test all poitns for now. Optimize later.
		if (dist < ljInfoVecs.Rcutoff) {
			tempIds.push_back(i);
		}
	}
	ljInfoVecs.node_id_close.resize( tempIds.size() );
	thrust::copy(tempIds.begin(), tempIds.end(), ljInfoVecs.node_id_close.begin());
	std::cout<<"lj nodes: "<< ljInfoVecs.node_id_close.size() << std::endl;*/






	//last, set memory foor buckets.
	auxVecs.id_bucket.resize(generalParams.maxNodeCount);
	auxVecs.id_value.resize(generalParams.maxNodeCount);
	auxVecs.id_bucket_expanded.resize(27 * (generalParams.maxNodeCount));
	auxVecs.id_value_expanded.resize(27 *( generalParams.maxNodeCount ));
 


};


